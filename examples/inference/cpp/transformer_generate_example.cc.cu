#include "hip/hip_runtime.h"
#include <algorithm>

#include "decoder.h"
#include "encoder.h"
#include "util.h"

/**
@file
Example of how to run transformer generation inference using our implementation.
*/

// Appoint precision.
const lightseq::cuda::OperationType optype =
    lightseq::cuda::OperationType::FP32;

int main(int argc, char *argv[]) {
  /* ---step1. init environment--- */
  hipStream_t stream_;
  hipblasHandle_t hd_;
  hipSetDevice(0);
  hipStreamCreate(&stream_);
  hipblasCreate(&hd_);
  hipblasSetStream(hd_, stream_);
  typedef lightseq::cuda::OperationTypeTraits<optype> optraits;

  /* ---step2. load model weights into GPU memory--- */
  lightseq::cuda::TransformerWeight<optype> tw_;
  // saved in custom proto file
  std::string model_weights_path = argv[1];
  std::string res = tw_.initializing(model_weights_path);
  if (!res.empty()) {
    std::cout << res << std::endl;
    return 0;
  }
  // when use sampling methods for decoding, set beam_size=1 to save memory
  if (tw_._sampling_method != "beam_search") {
    tw_._beam_size = 1;
  }
  tw_.print_model_config();

  /*
    step3. instantiate encoder and decoder, init the gpu memory buffer.
      using thrust vector to avoid manage gpu memory by hand
  */
  // instantiate encoder
  int max_batch_size = 8;
  thrust::device_vector<int> d_input_ =
      std::vector<int>(max_batch_size * tw_._max_step, 0);
  thrust::device_vector<int> d_padding_mask_ =
      std::vector<int>(max_batch_size * tw_._max_step, 0);
  thrust::device_vector<int> d_encoder_output_ =
      std::vector<int>(max_batch_size * tw_._max_step * tw_._hidden_size, 0);
  thrust::device_vector<int> d_output_ =
      std::vector<int>(max_batch_size * tw_._beam_size * tw_._max_step, 0);
  std::shared_ptr<lightseq::cuda::Encoder<optype>> encoder_ =
      std::make_shared<lightseq::cuda::Encoder<optype>>(
          max_batch_size,
          reinterpret_cast<int *>(thrust::raw_pointer_cast(d_input_.data())),
          reinterpret_cast<int *>(
              thrust::raw_pointer_cast(d_padding_mask_.data())),
          reinterpret_cast<optraits::DataType *>(
              thrust::raw_pointer_cast(d_encoder_output_.data())),
          tw_, stream_, hd_);
  res = encoder_->check();
  if (!res.empty()) {
    std::cout << res << std::endl;
    return 1;
  }
  // instantiate decoder
  std::shared_ptr<lightseq::cuda::Decoder<optype>> decoder_ =
      std::make_shared<lightseq::cuda::Decoder<optype>>(
          max_batch_size,
          reinterpret_cast<int *>(
              thrust::raw_pointer_cast(d_padding_mask_.data())),
          reinterpret_cast<optraits::DataType *>(
              thrust::raw_pointer_cast(d_encoder_output_.data())),
          reinterpret_cast<int *>(thrust::raw_pointer_cast(d_output_.data())),
          tw_, stream_, hd_, true);
  res = decoder_->check();
  if (!res.empty()) {
    std::cout << res << std::endl;
    return 1;
  }
  // init gpu memory buffer
  long buf_bytesize = std::max(encoder_->compute_buffer_bytesize(),
                               decoder_->compute_buffer_bytesize());
  // thrust::device_vector<int> d_buf_ =
  //     std::vector<int>(buf_bytesize / sizeof(int), 0);
  void *d_buf_;
  // encoder and decoder use the same buffer to save gpu memory useage
  lightseq::cuda::CHECK_GPU_ERROR(hipMalloc(&d_buf_, buf_bytesize));
  // encoder and decoder use the same buffer to save gpu memory useage
  encoder_->init_buffer(d_buf_);
  decoder_->init_buffer(d_buf_);
  hipStreamSynchronize(stream_);

  /* ---step4. read input token ids from file--- */
  int batch_size;
  int batch_seq_len;
  std::vector<int> host_input;
  // the first line of input file should
  // be two integers: batch_size and batch_seq_len.
  // followed by batch_size lines of
  // batch_seq_len integers, e.g.
  // 2 3
  // 666 666 666
  // 666 666 666
  std::string input_file_name = argv[2];
  lightseq::cuda::read_batch_tokenids_from_file(input_file_name, batch_size,
                                                batch_seq_len, host_input);

  /* ---step5. infer and log--- */
  auto start = std::chrono::high_resolution_clock::now();
  int sum_sample_step = 0;
  for (int i = 0; i < 1; i++) {
    // copy inputs from cpu memory to gpu memory
    hipMemcpyAsync(
        reinterpret_cast<int *>(thrust::raw_pointer_cast(d_input_.data())),
        host_input.data(), sizeof(int) * batch_size * batch_seq_len,
        hipMemcpyHostToDevice, stream_);
    encoder_->run_one_infer(batch_size, batch_seq_len);
    decoder_->run_one_infer(batch_size, batch_seq_len);
    sum_sample_step += decoder_->_cur_step;
    for (int ii = 0; ii < batch_size; ii++) {
      for (int j = 0; j < tw_._beam_size; j++) {
        lightseq::cuda::print_vec(
            d_output_.data() + ii * tw_._beam_size * (decoder_->_cur_step + 1) +
                j * (decoder_->_cur_step + 1),
            "Beam result", decoder_->_cur_step + 1);
      }
    }
  }
  lightseq::cuda::print_time_duration(start, "Infer time", stream_);
  std::cout << "Total sampled steps: " << sum_sample_step << std::endl;
  return 0;
}
