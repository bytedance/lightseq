#include "hip/hip_runtime.h"
#include <algorithm>

#include "decoder.h"
#include "encoder.h"
#include "util.h"

/**
@file
Example of how to run transformer inference using our implementation.
*/

// Appoint precision.
#ifdef FP16_MODE
const lightseq::cuda::OperationType OPTYPE =
    lightseq::cuda::OperationType::FP16;
#else
const lightseq::cuda::OperationType OPTYPE =
    lightseq::cuda::OperationType::FP32;
#endif

int main(int argc, char *argv[]) {
  /* ---step1. init environment--- */
  hipStream_t stream_;
  hipblasHandle_t hd_;
  hipSetDevice(0);
  hipStreamCreate(&stream_);
  hipblasCreate(&hd_);
  hipblasSetStream(hd_, stream_);
  typedef lightseq::cuda::OperationTypeTraits<OPTYPE> optraits;

  /* ---step2. load model weights into GPU memory--- */
  lightseq::cuda::TransformerWeight<OPTYPE> tw_;
  // saved in custom proto file
  std::string model_weights_path = argv[1];
  std::string res = tw_.initializing(model_weights_path);
  if (!res.empty()) {
    std::cout << res << std::endl;
    return 0;
  }

  /*
    step3. instantiate encoder and decoder, init the gpu memory buffer.
      using thrust vector to avoid manage gpu memory by hand
  */
  // instantiate encoder
  int max_batch_size = 8;
  thrust::device_vector<int> d_input_ =
      std::vector<int>(max_batch_size * tw_._max_step, 0);
  thrust::device_vector<int> d_padding_mask_ =
      std::vector<int>(max_batch_size * tw_._max_step, 0);
  thrust::device_vector<int> d_encoder_output_ =
      std::vector<int>(max_batch_size * tw_._max_step * tw_._hidden_size, 0);
  thrust::device_vector<int> d_output_ =
      std::vector<int>(max_batch_size * tw_._max_step, 0);
  std::shared_ptr<lightseq::cuda::Encoder<OPTYPE>> encoder_ =
      std::make_shared<lightseq::cuda::Encoder<OPTYPE>>(
          max_batch_size,
          reinterpret_cast<int *>(thrust::raw_pointer_cast(d_input_.data())),
          reinterpret_cast<int *>(
              thrust::raw_pointer_cast(d_padding_mask_.data())),
          reinterpret_cast<optraits::DataType *>(
              thrust::raw_pointer_cast(d_encoder_output_.data())),
          tw_, stream_, hd_);
  res = encoder_->check();
  if (!res.empty()) {
    std::cout << res << std::endl;
    return 1;
  }
  // instantiate decoder
  std::shared_ptr<lightseq::cuda::Decoder<OPTYPE>> decoder_ =
      std::make_shared<lightseq::cuda::Decoder<OPTYPE>>(
          max_batch_size,
          reinterpret_cast<int *>(
              thrust::raw_pointer_cast(d_padding_mask_.data())),
          reinterpret_cast<optraits::DataType *>(
              thrust::raw_pointer_cast(d_encoder_output_.data())),
          reinterpret_cast<int *>(thrust::raw_pointer_cast(d_output_.data())),
          tw_, stream_, hd_, false,
          reinterpret_cast<int *>(thrust::raw_pointer_cast(d_input_.data())));
  res = decoder_->check();
  if (!res.empty()) {
    std::cout << res << std::endl;
    return 1;
  }
  // init gpu memory buffer
  long buf_bytesize = std::max(encoder_->compute_buffer_bytesize(),
                               decoder_->compute_buffer_bytesize());
  thrust::device_vector<int> d_buf_ =
      std::vector<int>(buf_bytesize / sizeof(int), 0);
  // encoder and decoder use the same buffer to save gpu memory useage
  encoder_->init_buffer(
      reinterpret_cast<void *>(thrust::raw_pointer_cast(d_buf_.data())));
  decoder_->init_buffer(
      reinterpret_cast<void *>(thrust::raw_pointer_cast(d_buf_.data())));
  hipStreamSynchronize(stream_);

  /* ---step4. read input token ids from file--- */
  int batch_size;
  int batch_seq_len;
  std::vector<int> host_input;
  // the first line of input file should
  // be two integers: batch_size and batch_seq_len.
  // followed by batch_size lines of
  // batch_seq_len integers, e.g.
  // 2 3
  // 666 666 666
  // 666 666 666
  std::string input_file_name = argv[2];
  lightseq::cuda::read_batch_tokenids_from_file(input_file_name, batch_size,
                                                batch_seq_len, host_input);

  /* ---step5. infer and log--- */
  for (int i = 0; i < 1; i++) {
    auto start = std::chrono::high_resolution_clock::now();
    // copy inputs from cpu memory to gpu memory
    hipMemcpyAsync(
        reinterpret_cast<int *>(thrust::raw_pointer_cast(d_input_.data())),
        host_input.data(), sizeof(int) * batch_size * batch_seq_len,
        hipMemcpyHostToDevice, stream_);
    encoder_->run_one_infer(batch_size, batch_seq_len);
    decoder_->run_one_infer(batch_size, batch_seq_len);
    lightseq::cuda::print_time_duration(start, "one infer time", stream_);
    for (int ii = 0; ii < batch_size; ii++) {
      lightseq::cuda::print_vec(
          d_output_.data() + ii * (decoder_->_cur_step + 1), "finial res",
          decoder_->_cur_step + 1);
    }
  }
  return 0;
}
