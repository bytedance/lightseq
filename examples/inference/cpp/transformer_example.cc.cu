#include "hip/hip_runtime.h"
#include <algorithm>

#include "decoder.h"
#include "encoder.h"
#include "util.h"
#include "../kernels/embKernels.h"

/**
@file
Example of how to run transformer inference using our implementation.
*/

// Appoint precision.
#ifdef FP16_MODE
const lightseq::cuda::OperationType OPTYPE =
    lightseq::cuda::OperationType::FP16;
#else
const lightseq::cuda::OperationType OPTYPE =
    lightseq::cuda::OperationType::FP32;
#endif

int main(int argc, char *argv[]) {
  /* ---step1. init environment--- */
  hipStream_t stream_;
  hipblasHandle_t hd_;
  hipSetDevice(0);
  hipStreamCreate(&stream_);
  hipblasCreate(&hd_);
  hipblasSetStream(hd_, stream_);
  typedef lightseq::cuda::OperationTypeTraits<OPTYPE> optraits;

  /* ---step2. load model weights into GPU memory--- */
  lightseq::cuda::TransformerWeight<OPTYPE> tw_;
  // saved in custom proto file
  std::string model_weights_path = argv[1];
  std::string res = tw_.initializing(model_weights_path);
  if (!res.empty()) {
    std::cout << res << std::endl;
    return 0;
  }
  tw_.print_model_config();

  /*
    step3. instantiate encoder and decoder, init the gpu memory buffer.
      using thrust vector to avoid manage gpu memory by hand
  */
  // instantiate encoder
  int max_batch_size = 8;
  thrust::device_vector<int> d_input_ =
      std::vector<int>(max_batch_size * tw_._max_step, 0);
  thrust::device_vector<int> d_input_copy_ =
      std::vector<int>(max_batch_size * tw_._max_step, 0);
  thrust::device_vector<int> d_padding_mask_ =
      std::vector<int>(max_batch_size * tw_._max_step, 0);
  thrust::device_vector<int> d_encoder_output_ =
      std::vector<int>(max_batch_size * tw_._max_step * tw_._hidden_size, 0);
  thrust::device_vector<int> d_output_ =
      std::vector<int>(max_batch_size * tw_._max_step, 0);
  thrust::device_vector<int> d_src_lang_id_ =
      std::vector<int>(max_batch_size, 0);
  thrust::device_vector<int> d_trg_lang_id_ =
      std::vector<int>(max_batch_size, 0);

  int *p_d_input_ = (int *)(thrust::raw_pointer_cast(d_input_.data()));
  int *p_d_input_copy_ =
      (int *)(thrust::raw_pointer_cast(d_input_copy_.data()));
  int *p_d_padding_mask_ =
      (int *)(thrust::raw_pointer_cast(d_padding_mask_.data()));
  optraits::DataType *p_d_encoder_output_ =
      (optraits::DataType *)(thrust::raw_pointer_cast(
          d_encoder_output_.data()));
  int *p_d_output_ = (int *)(thrust::raw_pointer_cast(d_output_.data()));
  int *p_d_src_lang_id_ =
      (int *)(thrust::raw_pointer_cast(d_src_lang_id_.data()));
  int *p_d_trg_lang_id_ =
      (int *)(thrust::raw_pointer_cast(d_trg_lang_id_.data()));

  // instantiate encoder
  std::shared_ptr<lightseq::cuda::Encoder<OPTYPE>> encoder_ =
      std::make_shared<lightseq::cuda::Encoder<OPTYPE>>(
          max_batch_size, p_d_input_, p_d_padding_mask_, p_d_encoder_output_,
          tw_, stream_, hd_, p_d_src_lang_id_);
  res = encoder_->check();
  if (!res.empty()) {
    std::cout << res << std::endl;
    return 1;
  }

  // instantiate decoder
  std::shared_ptr<lightseq::cuda::Decoder<OPTYPE>> decoder_ =
      std::make_shared<lightseq::cuda::Decoder<OPTYPE>>(
          max_batch_size, p_d_padding_mask_, p_d_encoder_output_, p_d_output_,
          tw_, stream_, hd_, false, p_d_trg_lang_id_);
  res = decoder_->check();
  if (!res.empty()) {
    std::cout << res << std::endl;
    return 1;
  }

  // init gpu memory buffer
  long buf_bytesize = std::max(encoder_->compute_buffer_bytesize(),
                               decoder_->compute_buffer_bytesize());
  thrust::device_vector<int> d_buf_ =
      std::vector<int>(buf_bytesize / sizeof(int), 0);
  // encoder and decoder use the same buffer to save gpu memory useage
  encoder_->init_buffer(
      reinterpret_cast<void *>(thrust::raw_pointer_cast(d_buf_.data())));
  decoder_->init_buffer(
      reinterpret_cast<void *>(thrust::raw_pointer_cast(d_buf_.data())));
  hipStreamSynchronize(stream_);

  /* ---step4. read input token ids from file--- */
  int batch_size;
  int batch_seq_len;
  std::vector<int> host_input;
  // the first line of input file should
  // be two integers: batch_size and batch_seq_len.
  // followed by batch_size lines of
  // batch_seq_len integers, e.g.
  // 2 3
  // 666 666 666
  // 666 666 666
  std::string input_file_name = argv[2];
  lightseq::cuda::read_batch_tokenids_from_file(input_file_name, batch_size,
                                                batch_seq_len, host_input);

  /* ---step5. infer and log--- */
  for (int i = 0; i < 1; i++) {
    auto start = std::chrono::high_resolution_clock::now();
    // copy inputs from cpu memory to gpu memory
    hipMemcpyAsync(tw_._multilg_type == 0 ? p_d_input_ : p_d_input_copy_,
                    host_input.data(), sizeof(int) * batch_size * batch_seq_len,
                    hipMemcpyHostToDevice, stream_);
    if (tw_._multilg_type != 0) {
      // multilg request: src_lang_id, trg_lang_id, src_token0, src_token1...
      lightseq::cuda::launch_split_multilg_request(
          p_d_input_copy_, p_d_src_lang_id_, p_d_trg_lang_id_, p_d_input_,
          batch_size, batch_seq_len, stream_);
    }
    if (tw_._multilg_type == 1) {
      batch_seq_len -= 2;
    }
    if (tw_._multilg_type == 2) {
      batch_seq_len -= 1;
    }

    encoder_->run_one_infer(batch_size, batch_seq_len);
    decoder_->run_one_infer(batch_size, batch_seq_len);
    lightseq::cuda::print_time_duration(start, "one infer time", stream_);
    for (int ii = 0; ii < batch_size; ii++) {
      lightseq::cuda::print_vec(
          d_output_.data() + ii * (decoder_->_cur_step + 1), "finial res",
          decoder_->_cur_step + 1);
    }
  }
  return 0;
}
