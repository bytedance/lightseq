#include "hip/hip_runtime.h"
#include <math.h>

#include <cub/block/block_load.cuh>
#include <hipcub/hipcub.hpp>

#include "block_reduce.h"
#include "kernels.h"
#include "cuda_util.h"

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;
const float EPSILON = 1e-8f;

/**
@brief: softmax_kernel
Softmax forward kernel for
  enc-self-attn, dec-self-attn, encdec-attn

@thread
gridDim.x = dynamic
gridDim.y = batch_size
gridDim.z = nhead
blockDim.x = from_len

@param
inp: [batch_size, nhead, from_len, to_len], softmax input.
attn_mask: [batch_size, to_len], padding tokens are -inf,
  non padding tokens are 0.
  attn_mask!=nullptr for enc-self-attn and enc-dec-attn
  attn_mask=nullptr and mask_future=ture for dec-self-attn training
  attn_mask=nullptr and mask_future=false for dec-self-attn infer
*/
template <typename T, int block_dim, int ele_per_thread>
__global__ void ker_attn_softmax(T *out, T *inp, const T *attn_mask,
                                 int from_len, int to_len, bool mask_future) {
  int batch_id = blockIdx.y;
  int head_id = blockIdx.z;
  const int nhead = gridDim.z;
  const int token_per_reduce = 1;
  typedef hipcub::BlockLoad<T, block_dim, ele_per_thread,
                         hipcub::BLOCK_LOAD_VECTORIZE>
      BlockLoad;
  __shared__ typename BlockLoad::TempStorage ts_load;
  typedef hipcub::BlockStore<T, block_dim, ele_per_thread,
                          hipcub::BLOCK_STORE_VECTORIZE>
      BlockStore;
  __shared__ typename BlockStore::TempStorage ts_store;

  T mval[ele_per_thread];
  if (attn_mask) {
    attn_mask += batch_id * to_len;
    BlockLoad(ts_load).Load(attn_mask, mval, to_len, REDUCE_FLOAT_INF_NEG);
  }

  inp += flat_3dim(batch_id, head_id, 0, nhead, from_len * to_len);
  out += flat_3dim(batch_id, head_id, 0, nhead, from_len * to_len);
  for (int token_id = blockIdx.x * token_per_reduce; token_id < from_len;
       token_id += gridDim.x * token_per_reduce) {
    T inp_val[token_per_reduce][ele_per_thread];
    for (int i = 0; i < token_per_reduce && (token_id + i) < from_len; i++) {
      BlockLoad(ts_load).Load(inp + (token_id + i) * to_len, inp_val[i], to_len,
                              REDUCE_FLOAT_INF_NEG);
    }

    /* step 1. compute max */
    // thread local max
    float val[token_per_reduce][ele_per_thread];
    float l_max[token_per_reduce];
    for (int i = 0; i < token_per_reduce; i++) {
      l_max[i] = REDUCE_FLOAT_INF_NEG;
      for (int j = 0; j < ele_per_thread; j++) {
        float temp_val;
        if (mask_future && ele_per_thread * threadIdx.x + j > token_id + i) {
          temp_val = REDUCE_FLOAT_INF_NEG;
        } else {
          temp_val = (float)inp_val[i][j];
          if (attn_mask) {
            temp_val += (float)mval[j];
          }
        }
        val[i][j] = temp_val;
        l_max[i] = fmaxf(l_max[i], temp_val);
      }
    }
    // block reduce max
    blockReduce<ReduceType::kMax, token_per_reduce>(l_max);
    // write shared
    __shared__ float s_max[token_per_reduce];
    if (threadIdx.x == 0) {
      for (int i = 0; i < token_per_reduce; i++) {
        s_max[i] = l_max[i];
      }
    }
    __syncthreads();

    /* step 2. compute sum */
    // thread local sum
    float l_sum[token_per_reduce];
    for (int i = 0; i < token_per_reduce; i++) {
      l_sum[i] = 0.f;
      for (int j = 0; j < ele_per_thread; j++) {
        val[i][j] = __expf(val[i][j] - s_max[i]);
        l_sum[i] += val[i][j];
      }
    }
    // block reduce sum
    blockReduce<ReduceType::kSum, token_per_reduce>(l_sum);
    // write shared
    __shared__ float s_sum[token_per_reduce];
    if (threadIdx.x == 0) {
      for (int i = 0; i < token_per_reduce; i++) {
        s_sum[i] = __fdividef(1.0f, l_sum[i] + EPSILON);
      }
    }
    __syncthreads();

    /* step 3. compute final result */
    for (int i = 0; i < token_per_reduce && (token_id + i) < from_len; i++) {
      for (int j = 0; j < ele_per_thread; j++) {
        inp_val[i][j] = (T)(val[i][j] * s_sum[i]);
      }
      BlockStore(ts_store).Store(out + (token_id + i) * to_len, inp_val[i],
                                 to_len);
    }
  }  // blockIdx.x
}

template <typename T, int block_dim, int ele_per_thread>
__global__ void ker_attn_softmax_lt32(T *out, T *inp, const T *attn_mask,
                                      int from_len, int to_len,
                                      bool mask_future) {
  int batch_id = blockIdx.y;
  int head_id = blockIdx.z;
  const int nhead = gridDim.z;
  const int token_per_reduce = 1;
  typedef hipcub::BlockLoad<T, block_dim, ele_per_thread,
                         hipcub::BLOCK_LOAD_VECTORIZE>
      BlockLoad;
  __shared__ typename BlockLoad::TempStorage ts_load;
  typedef hipcub::BlockStore<T, block_dim, ele_per_thread,
                          hipcub::BLOCK_STORE_VECTORIZE>
      BlockStore;
  __shared__ typename BlockStore::TempStorage ts_store;

  T mval[ele_per_thread];
  if (attn_mask) {
    attn_mask += batch_id * to_len;
    BlockLoad(ts_load).Load(attn_mask, mval, to_len, REDUCE_FLOAT_INF_NEG);
  }

  inp += flat_3dim(batch_id, head_id, 0, nhead, from_len * to_len);
  out += flat_3dim(batch_id, head_id, 0, nhead, from_len * to_len);
  for (int token_id = blockIdx.x * token_per_reduce; token_id < from_len;
       token_id += gridDim.x * token_per_reduce) {
    T inp_val[token_per_reduce][ele_per_thread];
    for (int i = 0; i < token_per_reduce && (token_id + i) < from_len; i++) {
      BlockLoad(ts_load).Load(inp + (token_id + i) * to_len, inp_val[i], to_len,
                              REDUCE_FLOAT_INF_NEG);
    }

    /* step 1. compute max */
    // thread local max
    float val[token_per_reduce][ele_per_thread];
    float l_max[token_per_reduce];
    for (int i = 0; i < token_per_reduce; i++) {
      l_max[i] = REDUCE_FLOAT_INF_NEG;
      for (int j = 0; j < ele_per_thread; j++) {
        float temp_val;
        if (mask_future && ele_per_thread * threadIdx.x + j > token_id + i) {
          temp_val = REDUCE_FLOAT_INF_NEG;
        } else {
          temp_val = (float)inp_val[i][j];
          if (attn_mask) {
            temp_val += (float)mval[j];
          }
        }
        val[i][j] = temp_val;
        l_max[i] = fmaxf(l_max[i], temp_val);
      }
    }
    // warp reduce max
    warpReduce<ReduceType::kMax, token_per_reduce>(l_max);

    /* step 2. compute sum */
    // thread local sum
    float l_sum[token_per_reduce];
    for (int i = 0; i < token_per_reduce; i++) {
      l_sum[i] = 0.f;
      for (int j = 0; j < ele_per_thread; j++) {
        val[i][j] = __expf(val[i][j] - l_max[i]);
        l_sum[i] += val[i][j];
      }
    }
    // warp reduce sum
    warpReduce<ReduceType::kSum, token_per_reduce>(l_sum);

    /* step 3. compute final result */
    for (int i = 0; i < token_per_reduce && (token_id + i) < from_len; i++) {
      l_sum[i] = __fdividef(1.0f, l_sum[i] + EPSILON);
      for (int j = 0; j < ele_per_thread; j++) {
        inp_val[i][j] = (T)(val[i][j] * l_sum[i]);
      }
      BlockStore(ts_store).Store(out + (token_id + i) * to_len, inp_val[i],
                                 to_len);
    }
  }  // blockIdx.x
}

/*
  attn_mask!=nullptr for enc-self-attn and enc-dec-attn
  attn_mask=nullptr and mask_future=ture for dec-self-attn training
  attn_mask=nullptr and mask_future=false for dec-self-attn infer
*/
template <>
void launch_attn_softmax<float>(float *out, float *inp, const float *attn_mask,
                                int batch_size, int nhead, int from_len,
                                int to_len, bool mask_future,
                                hipStream_t stream) {
  dim3 grid_dim(1, batch_size, nhead);
  if (to_len <= 32) {
    ker_attn_softmax_lt32<float, 32, 1><<<grid_dim, 32, 0, stream>>>(
        out, inp, attn_mask, from_len, to_len, mask_future);
  } else if (to_len <= 64) {
    ker_attn_softmax_lt32<float, 32, 2><<<grid_dim, 32, 0, stream>>>(
        out, inp, attn_mask, from_len, to_len, mask_future);
  } else if (to_len <= 128) {
    grid_dim.x = 16;
    ker_attn_softmax<float, 64, 2><<<grid_dim, 64, 0, stream>>>(
        out, inp, attn_mask, from_len, to_len, mask_future);
  } else if (to_len <= 256) {
    grid_dim.x = 32;
    ker_attn_softmax<float, 128, 2><<<grid_dim, 128, 0, stream>>>(
        out, inp, attn_mask, from_len, to_len, mask_future);
  } else if (to_len <= 512) {
    grid_dim.x = 64;
    ker_attn_softmax<float, 256, 2><<<grid_dim, 256, 0, stream>>>(
        out, inp, attn_mask, from_len, to_len, mask_future);
  } else if (to_len <= 1024) {
    grid_dim.x = 128;
    ker_attn_softmax<float, 512, 2><<<grid_dim, 512, 0, stream>>>(
        out, inp, attn_mask, from_len, to_len, mask_future);
  } else {
    throw std::runtime_error(
        "Sequence length greater than 512 is currently not supported");
  }
}

template <>
void launch_attn_softmax<__half>(__half *out, __half *inp,
                                 const __half *attn_mask, int batch_size,
                                 int nhead, int from_len, int to_len,
                                 bool mask_future, hipStream_t stream) {
  dim3 grid_dim(1, batch_size, nhead);
  if (to_len <= 32) {
    ker_attn_softmax_lt32<__half, 32, 1><<<grid_dim, 32, 0, stream>>>(
        out, inp, attn_mask, from_len, to_len, mask_future);
  } else if (to_len <= 64) {
    ker_attn_softmax_lt32<__half, 32, 2><<<grid_dim, 32, 0, stream>>>(
        out, inp, attn_mask, from_len, to_len, mask_future);
  } else if (to_len <= 128) {
    grid_dim.x = 8;
    ker_attn_softmax<__half, 64, 2><<<grid_dim, 64, 0, stream>>>(
        out, inp, attn_mask, from_len, to_len, mask_future);
  } else if (to_len <= 256) {
    grid_dim.x = 16;
    ker_attn_softmax<__half, 128, 2><<<grid_dim, 128, 0, stream>>>(
        out, inp, attn_mask, from_len, to_len, mask_future);
  } else if (to_len <= 512) {
    grid_dim.x = 32;
    ker_attn_softmax<__half, 256, 2><<<grid_dim, 256, 0, stream>>>(
        out, inp, attn_mask, from_len, to_len, mask_future);
  } else if (to_len <= 1024) {
    grid_dim.x = 64;
    ker_attn_softmax<__half, 512, 2><<<grid_dim, 512, 0, stream>>>(
        out, inp, attn_mask, from_len, to_len, mask_future);
  } else {
    throw std::runtime_error(
        "Sequence length greater than 512 is currently not supported");
  }
}
