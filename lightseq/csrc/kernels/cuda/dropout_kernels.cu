#include "hip/hip_runtime.h"
#include <chrono>
#include <ctime>

#include "kernels.h"

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;
namespace lightseq {
namespace cuda {

hiprandStatePhilox4_32_10_t *curandstate;

/**
 * @brief element-wise activation function on device, like Relu, Gelu
 *
 * @tparam enum class ActivationType, kRelu, kGelu
 * @tparam input type
 * @param any shape of float and __half2
 * @return same shape and type with input
 */
template <ActivationType, typename T>
__forceinline__ __device__ T activation_kernel(T x);

template <>
__device__ float activation_kernel<ActivationType::kGelu, float>(float x) {
  float cdf =
      0.5f *
      (1.0f + tanhf((0.7978845608028654f * (x + 0.044715f * x * x * x))));
  return x * cdf;
}

template <>
__device__ __half2
activation_kernel<ActivationType::kGelu, __half2>(__half2 val) {
  __half2 val_pow3 = __hmul2(val, __hmul2(val, val));
  float2 tmp_pow = __half22float2(val_pow3);
  float2 tmp = __half22float2(val);

  tmp.x =
      0.5f *
      (1.0f + tanhf((0.7978845608028654f * (tmp.x + 0.044715f * tmp_pow.x))));
  tmp.y =
      0.5f *
      (1.0f + tanhf((0.7978845608028654f * (tmp.y + 0.044715f * tmp_pow.y))));
  return __hmul2(val, __float22half2_rn(tmp));
}

template <>
__device__ float activation_kernel<ActivationType::kRelu, float>(float x) {
  return fmaxf(x, 0);
}

template <>
__device__ __half2
activation_kernel<ActivationType::kRelu, __half2>(__half2 x) {
  return __floats2half2_rn(fmaxf(0.f, __half2float(x.x)),
                           fmaxf(0.f, __half2float(x.y)));
}

/**
 * @brief element-wise activation backward function on device
 *
 * @tparam enum class ActivationType
 * @tparam input type
 * @param any shape of float and __half2
 * @return same shape of input
 */
template <ActivationType, typename T>
__forceinline__ __device__ T activation_bwd_kernel(T grad, T x);

template <>
__device__ float activation_bwd_kernel<ActivationType::kGelu, float>(float grad,
                                                                     float x) {
  const float sqrt_param = 0.79788456080286535587989211986876f;
  const float mul_param = 0.044715;

  float x2mul = x * x * mul_param;
  float tan_h = tanhf(sqrt_param * (x + x * x2mul));
  float dg1 = 0.5f * (1.0f + tan_h);
  float dg2 = x * 0.5f * sqrt_param * (1 - tan_h * tan_h);
  float dg3 = dg2 * 3 * x2mul;
  return grad * (dg1 + dg2 + dg3);
}

template <>
__device__ __half activation_bwd_kernel<ActivationType::kGelu, __half>(
    __half grad, __half x_half) {
  float x = __half2float(x_half);
  const float sqrt_param = 0.79788456080286535587989211986876f;
  const float mul_param = 0.044715;

  float x2mul = x * x * mul_param;
  float tan_h = tanhf(sqrt_param * (x + x * x2mul));
  float dg1 = 0.5f * (1.0f + tan_h);
  float dg2 = x * 0.5f * sqrt_param * (1 - tan_h * tan_h);
  float dg3 = dg2 * 3 * x2mul;
  return grad * __float2half(dg1 + dg2 + dg3);
}

template <>
__device__ float activation_bwd_kernel<ActivationType::kRelu, float>(float grad,
                                                                     float x) {
  return x > 0.f ? grad : 0.f;
}

template <>
__device__ __half
activation_bwd_kernel<ActivationType::kRelu, __half>(__half grad, __half x) {
  const __half half_zero = __float2half(0.f);
  return x > half_zero ? grad : half_zero;
}

template <>
__device__ __half2 activation_bwd_kernel<ActivationType::kRelu, __half2>(
    __half2 grad2, __half2 x_half2) {
  const __half half_zero = __float2half(0.f);
  return __floats2half2_rn(x_half2.x > half_zero ? grad2.x : half_zero,
                           x_half2.y > half_zero ? grad2.y : half_zero);
}

/**
 * @brief init hiprand states in global memory
 *
 * @thread grid_dim * block*dim to suuport any size of states
 * @param state persistant hiprand states
 * @param seed seed to init states
 * @return void
 */
__global__ void curand_init_kernel(hiprandStatePhilox4_32_10_t *state,
                                   int seed) {
  /* Each thread gets same seed, a different sequence
     number, no offset */
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(seed, id, 0, &state[id]);
}

void launch_curand_init(int total_count, int dim, hipStream_t stream) {
  hipMalloc(&curandstate, total_count * sizeof(hiprandStatePhilox4_32_10_t));
  int grid_dim = total_count >> 9;
  curand_init_kernel<<<grid_dim, 512, 0, stream>>>(
      curandstate, std::chrono::duration_cast<std::chrono::microseconds>(
                       std::chrono::system_clock::now().time_since_epoch())
                       .count());
}

/**
 * @brief element-wise dropout, store dropped position in mask, it's not
 * in-place
 *
 * @thread
 * gridDim.x = total_count / 1024
 * blockDim.x = 1024
 *
 * @param total_count total elements
 * @param ratio drop ratio
 * @param out any size of float and __half
 * @param in same with out
 * @param mask uint8 type, same size with out
 * @param seed seed to hiprand
 * @return void
 */
__global__ void ls_dropout_kernel(const int total_count, const float ratio,
                                  float *__restrict__ out,
                                  const float *__restrict__ in,
                                  uint8_t *__restrict__ mask, const int seed) {
  const float scale = 1.f / (1.f - ratio);
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 4 >= total_count) return;

  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, i, 0, &state);
  uint8_t m[4];

  float4 *out4 = reinterpret_cast<float4 *>(out);
  const float4 *data4 = reinterpret_cast<const float4 *>(in);
  uint32_t *mask4 = reinterpret_cast<uint32_t *>(mask);
  float4 rand = hiprand_uniform4(&state);

  m[0] = (uint8_t)(rand.x > ratio);
  m[1] = (uint8_t)(rand.y > ratio);
  m[2] = (uint8_t)(rand.z > ratio);
  m[3] = (uint8_t)(rand.w > ratio);

  uint32_t *m4 = reinterpret_cast<uint32_t *>(m);
  mask4[i] |= m4[0];

  float4 input4 = data4[i];
  float4 res4;
  res4.x = input4.x * scale * m[0];
  res4.y = input4.y * scale * m[1];
  res4.z = input4.z * scale * m[2];
  res4.w = input4.w * scale * m[3];
  out4[i] = res4;
}

__global__ void ls_dropout_kernel(const int total_count, const float ratio,
                                  __half *__restrict__ out,
                                  const __half *__restrict__ in,
                                  uint8_t *__restrict__ mask, const int seed) {
  const float scale = 1.f / (1.f - ratio);

  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 8 >= total_count) return;

  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, i, 0, &state);

  const float4 *vals_float4 = reinterpret_cast<const float4 *>(in);
  float4 *outs_float4 = reinterpret_cast<float4 *>(out);
  uint64_t *mask8 = reinterpret_cast<uint64_t *>(mask);

  uint8_t m[8];
  float4 rand = hiprand_uniform4(&state);
  m[0] = (uint8_t)(rand.x > ratio);
  m[1] = (uint8_t)(rand.y > ratio);
  m[2] = (uint8_t)(rand.z > ratio);
  m[3] = (uint8_t)(rand.w > ratio);
  rand = hiprand_uniform4(&state);
  m[4] = (uint8_t)(rand.x > ratio);
  m[5] = (uint8_t)(rand.y > ratio);
  m[6] = (uint8_t)(rand.z > ratio);
  m[7] = (uint8_t)(rand.w > ratio);
  uint64_t *m8 = reinterpret_cast<uint64_t *>(m);
  mask8[i] |= *m8;

  float4 val_float4 = vals_float4[i];
  float4 out_float4;
  __half2 *val_half2 = reinterpret_cast<__half2 *>(&val_float4);
  __half2 *out_half2 = reinterpret_cast<__half2 *>(&out_float4);
  __half2 scale_mask_1 = __floats2half2_rn(scale * m[0], scale * m[1]);
  __half2 scale_mask_2 = __floats2half2_rn(scale * m[2], scale * m[3]);
  __half2 scale_mask_3 = __floats2half2_rn(scale * m[4], scale * m[5]);
  __half2 scale_mask_4 = __floats2half2_rn(scale * m[6], scale * m[7]);
  out_half2[0] = __hmul2(val_half2[0], scale_mask_1);
  out_half2[1] = __hmul2(val_half2[1], scale_mask_2);
  out_half2[2] = __hmul2(val_half2[2], scale_mask_3);
  out_half2[3] = __hmul2(val_half2[3], scale_mask_4);
  outs_float4[i] = out_float4;
}

/**
 * @brief element-wise dropout backward with dropout mask, it's
 * not in-place
 *
 * @thread
 * gridDim.x = total_count / 1024
 * blockDim.x = 1024
 *
 * @param total_count total elements
 * @param ratio drop ratio
 * @param in any size of float and __half
 * @param mask uint8 type, same size with in
 * @return void
 */
__global__ void ls_dropout_bwd_kernel(const int total_count, const float ratio,
                                      float *out, const float *in,
                                      const uint8_t *__restrict__ mask) {
  const float scale = 1.f / (1.f - ratio);
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 4 >= total_count) return;

  uint8_t m[4];

  float4 *out4 = reinterpret_cast<float4 *>(out);
  const float4 *in4 = reinterpret_cast<const float4 *>(in);
  const uint32_t *mask4 = reinterpret_cast<const uint32_t *>(mask);

  uint32_t *m4 = reinterpret_cast<uint32_t *>(m);
  m4[0] = mask4[i];

  float4 input4 = in4[i];
  float4 res4;
  res4.x = input4.x * scale * static_cast<float>(m[0] & 1);
  res4.y = input4.y * scale * static_cast<float>(m[1] & 1);
  res4.z = input4.z * scale * static_cast<float>(m[2] & 1);
  res4.w = input4.w * scale * static_cast<float>(m[3] & 1);
  out4[i] = res4;
}

__global__ void ls_dropout_bwd_kernel(const int total_count, const float ratio,
                                      __half *out, const __half *in,
                                      const uint8_t *__restrict__ mask) {
  const __half scale = 1.f / (1.f - ratio);

  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 8 >= total_count) return;

  float4 *out4 = reinterpret_cast<float4 *>(out);
  const float4 *vals_float4 = reinterpret_cast<const float4 *>(in);
  const uint64_t *mask8 = reinterpret_cast<const uint64_t *>(mask);

  uint8_t m[8];
  uint64_t *m8 = reinterpret_cast<uint64_t *>(m);
  m8[0] = mask8[i];

  float4 val_float4 = vals_float4[i];
  float4 out_float4;
  __half2 *val_half2 = reinterpret_cast<__half2 *>(&val_float4);
  __half2 *out_half2 = reinterpret_cast<__half2 *>(&out_float4);
  __half2 scale_mask_1 = __halves2half2(scale * __float2half(m[0] & 1),
                                        scale * __float2half(m[1] & 1));
  __half2 scale_mask_2 = __halves2half2(scale * __float2half(m[2] & 1),
                                        scale * __float2half(m[3] & 1));
  __half2 scale_mask_3 = __halves2half2(scale * __float2half(m[4] & 1),
                                        scale * __float2half(m[5] & 1));
  __half2 scale_mask_4 = __halves2half2(scale * __float2half(m[6] & 1),
                                        scale * __float2half(m[7] & 1));
  out_half2[0] = __hmul2(val_half2[0], scale_mask_1);
  out_half2[1] = __hmul2(val_half2[1], scale_mask_2);
  out_half2[2] = __hmul2(val_half2[2], scale_mask_3);
  out_half2[3] = __hmul2(val_half2[3], scale_mask_4);
  out4[i] = out_float4;
}

template <>
void launch_ls_dropout<float>(float *out, const float *vals, uint8_t *mask,
                              int total_count, float ratio, hipStream_t stream,
                              bool backward) {
  int grid_dim = total_count >> 12;
  if (!backward) {
    ls_dropout_kernel<<<grid_dim + 1, 1024, 0, stream>>>(
        total_count, ratio, out, vals, mask,
        std::chrono::duration_cast<std::chrono::microseconds>(
            std::chrono::system_clock::now().time_since_epoch())
            .count());
  } else {
    ls_dropout_bwd_kernel<<<grid_dim + 1, 1024, 0, stream>>>(total_count, ratio,
                                                             out, vals, mask);
  }
}

template <>
void launch_ls_dropout<__half>(__half *out, const __half *vals, uint8_t *mask,
                               int total_count, float ratio,
                               hipStream_t stream, bool backward) {
  int grid_dim = total_count >> 13;
  if (!backward) {
    ls_dropout_kernel<<<grid_dim + 1, 1024, 0, stream>>>(
        total_count, ratio, out, vals, mask,
        std::chrono::duration_cast<std::chrono::microseconds>(
            std::chrono::system_clock::now().time_since_epoch())
            .count());
  } else {
    ls_dropout_bwd_kernel<<<grid_dim + 1, 1024, 0, stream>>>(total_count, ratio,
                                                             out, vals, mask);
  }
}

/**
 * @brief fused bias, dropout, and residual at the end of Attention and FFN,
 * store dropped position in mask, it's not in-place
 *
 * @thread
 * gridDim.x = total_count / 1024
 * blockDim.x = 1024
 *
 * @param total_count total elements
 * @param ratio drop ratio
 * @param out [batch_size, seq_len, hidden_size], float and __half
 * @param in [batch_size, seq_len, hidden_size], float and __half
 * @param mask [batch_size, seq_len, hidden_size], uint8 type
 * @param bias [hidden_size], ffn bias
 * @param residual [batch_size, seq_len, hidden_size], float and __half
 * @param seed seed to hiprand
 * @param hidden_size hidden size
 * @return void
 */
__global__ void ls_dropout_res_bias_kernel(
    const int total_count, const float ratio, float *__restrict__ out,
    const float *__restrict__ in, uint8_t *__restrict__ mask,
    const float *__restrict__ bias, const float *__restrict__ residual,
    const int seed, const int hidden_size) {
  const float scale = 1.f / (1.f - ratio);
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 4 >= total_count) return;

  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, i, 0, &state);
  uint8_t m[4];

  float4 *out4 = reinterpret_cast<float4 *>(out);
  const float4 *data4 = reinterpret_cast<const float4 *>(in);
  const float4 *residual4 = reinterpret_cast<const float4 *>(residual);
  const float4 *bias4 = reinterpret_cast<const float4 *>(bias);
  uint32_t *mask4 = reinterpret_cast<uint32_t *>(mask);
  float4 rand = hiprand_uniform4(&state);

  m[0] = static_cast<uint8_t>(rand.x > ratio);
  m[1] = static_cast<uint8_t>(rand.y > ratio);
  m[2] = static_cast<uint8_t>(rand.z > ratio);
  m[3] = static_cast<uint8_t>(rand.w > ratio);

  int bias_i = i % (hidden_size >> 2);
  uint32_t *m4 = reinterpret_cast<uint32_t *>(m);
  mask4[i] = m4[0];
  const float4 input4 = data4[i];
  const float4 b4 = __ldg(&bias4[bias_i]);
  const float4 res4 = residual4[i];
  float4 output4;

  output4.x = (input4.x + b4.x) * scale * m[0] + res4.x;
  output4.y = (input4.y + b4.y) * scale * m[1] + res4.y;
  output4.z = (input4.z + b4.z) * scale * m[2] + res4.z;
  output4.w = (input4.w + b4.w) * scale * m[3] + res4.w;

  out4[i] = output4;
}

__global__ void ls_dropout_res_bias_kernel(
    const int total_count, const float ratio, __half *__restrict__ out,
    const __half *__restrict__ in, uint8_t *__restrict__ mask,
    const __half *__restrict__ bias, const __half *__restrict__ residual,
    const int seed, const int hidden_size) {
  const __half scale = 1. / (1. - ratio);

  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 8 >= total_count) return;

  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, i, 0, &state);

  const float4 *vals_float4 = reinterpret_cast<const float4 *>(in);
  float4 *outs_float4 = reinterpret_cast<float4 *>(out);
  const float4 *residual4 = reinterpret_cast<const float4 *>(residual);
  const float4 *bias4 = reinterpret_cast<const float4 *>(bias);
  uint64_t *mask8 = reinterpret_cast<uint64_t *>(mask);

  uint8_t m[8];
  float4 rand = hiprand_uniform4(&state);
  m[0] = static_cast<uint8_t>(rand.x > ratio);
  m[1] = static_cast<uint8_t>(rand.y > ratio);
  m[2] = static_cast<uint8_t>(rand.z > ratio);
  m[3] = static_cast<uint8_t>(rand.w > ratio);
  rand = hiprand_uniform4(&state);
  m[4] = static_cast<uint8_t>(rand.x > ratio);
  m[5] = static_cast<uint8_t>(rand.y > ratio);
  m[6] = static_cast<uint8_t>(rand.z > ratio);
  m[7] = static_cast<uint8_t>(rand.w > ratio);
  uint64_t *m8 = reinterpret_cast<uint64_t *>(m);
  mask8[i] = m8[0];

  int bias_i = i % (hidden_size >> 3);
  float4 val_float4 = vals_float4[i];
  const float4 b4 = __ldg(&bias4[bias_i]);
  const float4 res4 = residual4[i];
  float4 out_float4;

  __half2 *val_half2 = reinterpret_cast<__half2 *>(&val_float4);
  __half2 *out_half2 = reinterpret_cast<__half2 *>(&out_float4);
  const __half2 *b_half2 = reinterpret_cast<const __half2 *>(&b4);
  const __half2 *res_half2 = reinterpret_cast<const __half2 *>(&res4);
  __half2 scale_mask_1 =
      __halves2half2(scale * __float2half(m[0]), scale * __float2half(m[1]));
  __half2 scale_mask_2 =
      __halves2half2(scale * __float2half(m[2]), scale * __float2half(m[3]));
  __half2 scale_mask_3 =
      __halves2half2(scale * __float2half(m[4]), scale * __float2half(m[5]));
  __half2 scale_mask_4 =
      __halves2half2(scale * __float2half(m[6]), scale * __float2half(m[7]));
  out_half2[0] =
      __hfma2(__hadd2(val_half2[0], b_half2[0]), scale_mask_1, res_half2[0]);
  out_half2[1] =
      __hfma2(__hadd2(val_half2[1], b_half2[1]), scale_mask_2, res_half2[1]);
  out_half2[2] =
      __hfma2(__hadd2(val_half2[2], b_half2[2]), scale_mask_3, res_half2[2]);
  out_half2[3] =
      __hfma2(__hadd2(val_half2[3], b_half2[3]), scale_mask_4, res_half2[3]);
  outs_float4[i] = out_float4;
}

template <>
void launch_ls_dropout_res_bias<float>(float *out, const float *vals,
                                       uint8_t *mask, const float *bias,
                                       const float *residual, int total_count,
                                       int dim, float ratio,
                                       hipStream_t stream) {
  int grid_dim = total_count >> 12;
  ls_dropout_res_bias_kernel<<<grid_dim + 1, 1024, 0, stream>>>(
      total_count, ratio, out, vals, mask, bias, residual,
      std::chrono::duration_cast<std::chrono::microseconds>(
          std::chrono::system_clock::now().time_since_epoch())
          .count(),
      dim);
}

template <>
void launch_ls_dropout_res_bias<__half>(__half *out, const __half *vals,
                                        uint8_t *mask, const __half *bias,
                                        const __half *residual, int total_count,
                                        int dim, float ratio,
                                        hipStream_t stream) {
  int grid_dim = total_count >> 13;
  ls_dropout_res_bias_kernel<<<grid_dim + 1, 1024, 0, stream>>>(
      total_count, ratio, out, vals, mask, bias, residual,
      std::chrono::duration_cast<std::chrono::microseconds>(
          std::chrono::system_clock::now().time_since_epoch())
          .count(),
      dim);
}

/**
 * @brief fused bias and dropout backward at the end of Attention and FFN
 *
 * @thread
 * gridDim.x = hidden_size / 8
 * blockDim.x = 8
 * blockDim.y = 1024 / 8 = 128
 *
 * @param row_size batch_size * seq_len
 * @param ratio dropout ratio
 * @param in_grad [batch_size, seq_len, hidden_size], input grad
 * @param bias_grad [hidden_size], bias grad
 * @param out_grad [batch_size, seq_len, hidden_size], output grad
 * @param mask [batch_size, seq_len, hidden_size], dropout mask
 * @param hidden_size
 * @return void
 */
__global__ void ls_dropout_bias_bwd_kernel(
    const int row_size, const float ratio, float *__restrict__ in_grad,
    float *__restrict__ bias_grad, const float *__restrict__ out_grad,
    const uint8_t *__restrict__ mask, const int hidden_size) {
  const float scale = 1.f / (1.f - ratio);
  // every block generate 8 bias result
  __shared__ float tile[8][129];

  cg::thread_block b = cg::this_thread_block();
  cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

  int col_idx = flat_2dim(blockIdx.x, threadIdx.x, 8);
  int stride = hidden_size * 128;
  float local_sum = 0;

  int idx = flat_2dim(threadIdx.y, col_idx, hidden_size);
  for (int r = threadIdx.y; r < row_size; r += 128) {
    float val = out_grad[idx];
    val *= scale * static_cast<float>(mask[idx] & 1);
    local_sum += val;
    in_grad[idx] = val;
    idx += stride;
  }

  tile[threadIdx.x][threadIdx.y] = local_sum;
  __syncthreads();

  float sum = 0;
  int tid = threadIdx.y * blockDim.x + threadIdx.x;
  int x = tid >> 7;
  int y = tid & (127);
  if (y < 32) {
#pragma unroll
    for (int i = 0; i < 4; i++) {
      sum += tile[x][y + i * 32];
    }
  }
  __syncthreads();

  for (int i = 1; i < 32; i <<= 1) sum += g.shfl_down(sum, i);

  if (y == 0) tile[0][x] = sum;
  __syncthreads();

  if (threadIdx.x < 8) {
    int pos = flat_2dim(blockIdx.x, threadIdx.x, 8);
    bias_grad[pos] = tile[0][threadIdx.x];
  }
}

__global__ void ls_dropout_bias_bwd_kernel(
    const int row_size, const float ratio, __half *__restrict__ in_grad,
    __half *__restrict__ bias_grad, const __half *__restrict__ out_grad,
    const uint8_t *__restrict__ mask, const int hidden_size) {
  const __half2 scale = __float2half2_rn(1.f / (1.f - ratio));
  __shared__ __half2 tile[8][129];

  cg::thread_block b = cg::this_thread_block();
  cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

  __half2 *in_grad2 = reinterpret_cast<__half2 *>(in_grad);
  const __half2 *out_grad2 = reinterpret_cast<const __half2 *>(out_grad);
  __half2 *bias_grad2 = reinterpret_cast<__half2 *>(bias_grad);

  int col_idx = flat_2dim(blockIdx.x, threadIdx.x, 8);
  int stride = hidden_size * 128;
  __half2 local_sum = __float2half2_rn(0.f);

  int idx = flat_2dim(threadIdx.y, col_idx, hidden_size);
  for (int r = threadIdx.y; r < row_size; r += 128) {
    __half2 val = out_grad2[idx];
    __half2 m2 = __floats2half2_rn(mask[2 * idx] & 1, mask[2 * idx + 1] & 1);
    val *= scale * m2;
    local_sum += val;
    in_grad2[idx] = val;
    idx += stride;
  }

  tile[threadIdx.x][threadIdx.y] = local_sum;
  __syncthreads();

  __half2 sum = __float2half2_rn(0.f);
  int tid = threadIdx.y * blockDim.x + threadIdx.x;
  int x = tid >> 7;
  int y = tid & (127);
  if (y < 32) {
#pragma unroll
    for (int i = 0; i < 4; i++) {
      sum += tile[x][y + i * 32];
    }
  }
  __syncthreads();

  for (int i = 1; i < WARP_SIZE; i <<= 1) sum += g.shfl_down(sum, i);

  if (y == 0) tile[0][x] = sum;
  __syncthreads();

  if (threadIdx.x < 8) {
    int pos = flat_2dim(blockIdx.x, threadIdx.x, 8);
    bias_grad2[pos] = tile[0][threadIdx.x];
  }
}

template <typename T>
void launch_ls_dropout_bias_bwd(T *in_grad, T *bias_grad, const T *out_grad,
                                const uint8_t *mask, int row_size, int dim,
                                float ratio, hipStream_t stream) {
  dim3 grid_dim((dim - 1) / 8 + 1);
  dim3 block_dim(8, 128);
  ls_dropout_bias_bwd_kernel<<<grid_dim, block_dim, 0, stream>>>(
      row_size, ratio, in_grad, bias_grad, out_grad, mask, dim);
}

template <>
void launch_ls_dropout_bias_bwd(__half *in_grad, __half *bias_grad,
                                const __half *out_grad, const uint8_t *mask,
                                int row_size, int dim, float ratio,
                                hipStream_t stream) {
  dim >>= 1;
  dim3 grid_dim((dim - 1) / 8 + 1);
  dim3 block_dim(8, 128);
  ls_dropout_bias_bwd_kernel<<<grid_dim, block_dim, 0, stream>>>(
      row_size, ratio, in_grad, bias_grad, out_grad, mask, dim);
}

template void launch_ls_dropout_bias_bwd(float *in_grad, float *bias_grad,
                                         const float *out_grad,
                                         const uint8_t *mask, int row_size,
                                         int dim, float ratio,
                                         hipStream_t stream);

/**
 * @brief fused bias, activation, and dropout at the end of first ffn
 *
 * @thread
 * gridDim.x = hidden_size / 8
 * blockDim.x = 8
 * blockDim.y = 1024 / 8 = 128
 *
 * @tparam act_type activation function, like kRelu, kGelu
 * @param total_count total elements
 * @param ratio drop ratio
 * @param out [batch_size, seq_len, hidden_size], float and __half
 * @param in [batch_size, seq_len, hidden_size], float and __half
 * @param mask [batch_size, seq_len, hidden_size], uint8 type
 * @param bias [hidden_size], ffn bias
 * @param seed seed to hiprand
 * @param hidden_size
 * @return void
 */
template <ActivationType act_type>
__global__ void ls_dropout_act_bias_kernel(
    const int total_count, const float ratio, float *__restrict__ out,
    const float *__restrict__ in, uint8_t *__restrict__ mask,
    const float *__restrict__ bias, const int seed, const int hidden_size) {
  const float scale = 1.f / (1.f - ratio);
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 4 >= total_count) return;

  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, i, 0, &state);
  uint8_t m[4];

  float4 *out4 = reinterpret_cast<float4 *>(out);
  const float4 *data4 = reinterpret_cast<const float4 *>(in);
  const float4 *bias4 = reinterpret_cast<const float4 *>(bias);
  uint32_t *mask4 = reinterpret_cast<uint32_t *>(mask);
  float4 rand = hiprand_uniform4(&state);

  m[0] = (uint8_t)(rand.x > ratio);
  m[1] = (uint8_t)(rand.y > ratio);
  m[2] = (uint8_t)(rand.z > ratio);
  m[3] = (uint8_t)(rand.w > ratio);

  int bias_i = i % (hidden_size >> 2);
  uint32_t *m4 = reinterpret_cast<uint32_t *>(m);
  mask4[i] = m4[0];
  const float4 input4 = data4[i];
  const float4 b4 = __ldg(&bias4[bias_i]);
  float4 output4;

  output4.x =
      activation_kernel<act_type, float>(input4.x + b4.x) * scale * m[0];
  output4.y =
      activation_kernel<act_type, float>(input4.y + b4.y) * scale * m[1];
  output4.z =
      activation_kernel<act_type, float>(input4.z + b4.z) * scale * m[2];
  output4.w =
      activation_kernel<act_type, float>(input4.w + b4.w) * scale * m[3];

  out4[i] = output4;
}

template <ActivationType act_type>
__global__ void ls_dropout_act_bias_kernel(
    const int total_count, const float ratio, __half *__restrict__ out,
    const __half *__restrict__ in, uint8_t *__restrict__ mask,
    const __half *__restrict__ bias, const int seed, const int hidden_size) {
  const float scale = 1.f / (1.f - ratio);

  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 8 >= total_count) return;

  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, i, 0, &state);

  const float4 *vals_float4 = reinterpret_cast<const float4 *>(in);
  float4 *outs_float4 = reinterpret_cast<float4 *>(out);
  const float4 *bias4 = reinterpret_cast<const float4 *>(bias);
  uint64_t *mask8 = reinterpret_cast<uint64_t *>(mask);

  uint8_t m[8];
  float4 rand = hiprand_uniform4(&state);
  m[0] = (uint8_t)(rand.x > ratio);
  m[1] = (uint8_t)(rand.y > ratio);
  m[2] = (uint8_t)(rand.z > ratio);
  m[3] = (uint8_t)(rand.w > ratio);
  rand = hiprand_uniform4(&state);
  m[4] = (uint8_t)(rand.x > ratio);
  m[5] = (uint8_t)(rand.y > ratio);
  m[6] = (uint8_t)(rand.z > ratio);
  m[7] = (uint8_t)(rand.w > ratio);
  uint64_t *m8 = reinterpret_cast<uint64_t *>(m);
  mask8[i] = *m8;

  int bias_i = i % (hidden_size >> 3);
  float4 val_float4 = vals_float4[i];
  const float4 b4 = __ldg(&bias4[bias_i]);
  float4 out_float4;

  __half2 *val_half2 = reinterpret_cast<__half2 *>(&val_float4);
  __half2 *out_half2 = reinterpret_cast<__half2 *>(&out_float4);
  const __half2 *b_half2 = reinterpret_cast<const __half2 *>(&b4);

  __half2 scale_mask_1 = __floats2half2_rn(scale * m[0], scale * m[1]);
  __half2 scale_mask_2 = __floats2half2_rn(scale * m[2], scale * m[3]);
  __half2 scale_mask_3 = __floats2half2_rn(scale * m[4], scale * m[5]);
  __half2 scale_mask_4 = __floats2half2_rn(scale * m[6], scale * m[7]);
  out_half2[0] = __hmul2(
      activation_kernel<act_type, __half2>(__hadd2(val_half2[0], b_half2[0])),
      scale_mask_1);
  out_half2[1] = __hmul2(
      activation_kernel<act_type, __half2>(__hadd2(val_half2[1], b_half2[1])),
      scale_mask_2);
  out_half2[2] = __hmul2(
      activation_kernel<act_type, __half2>(__hadd2(val_half2[2], b_half2[2])),
      scale_mask_3);
  out_half2[3] = __hmul2(
      activation_kernel<act_type, __half2>(__hadd2(val_half2[3], b_half2[3])),
      scale_mask_4);
  outs_float4[i] = out_float4;
}

template <>
void launch_ls_dropout_act_bias<ActivationType::kGelu, float>(
    float *out, const float *vals, uint8_t *mask, const float *bias,
    int total_count, int dim, float ratio, hipStream_t stream) {
  int grid_dim = total_count >> 10;
  ls_dropout_act_bias_kernel<ActivationType::kGelu>
      <<<grid_dim + 1, 256, 0, stream>>>(
          total_count, ratio, out, vals, mask, bias,
          std::chrono::duration_cast<std::chrono::microseconds>(
              std::chrono::system_clock::now().time_since_epoch())
              .count(),
          dim);
}

template <>
void launch_ls_dropout_act_bias<ActivationType::kGelu, __half>(
    __half *out, const __half *vals, uint8_t *mask, const __half *bias,
    int total_count, int dim, float ratio, hipStream_t stream) {
  int grid_dim = total_count >> 11;
  ls_dropout_act_bias_kernel<ActivationType::kGelu>
      <<<grid_dim + 1, 256, 0, stream>>>(
          total_count, ratio, out, vals, mask, bias,
          std::chrono::duration_cast<std::chrono::microseconds>(
              std::chrono::system_clock::now().time_since_epoch())
              .count(),
          dim);
}

template <>
void launch_ls_dropout_act_bias<ActivationType::kRelu, float>(
    float *out, const float *vals, uint8_t *mask, const float *bias,
    int total_count, int dim, float ratio, hipStream_t stream) {
  int grid_dim = total_count >> 10;
  ls_dropout_act_bias_kernel<ActivationType::kRelu>
      <<<grid_dim + 1, 256, 0, stream>>>(
          total_count, ratio, out, vals, mask, bias,
          std::chrono::duration_cast<std::chrono::microseconds>(
              std::chrono::system_clock::now().time_since_epoch())
              .count(),
          dim);
}

template <>
void launch_ls_dropout_act_bias<ActivationType::kRelu, __half>(
    __half *out, const __half *vals, uint8_t *mask, const __half *bias,
    int total_count, int dim, float ratio, hipStream_t stream) {
  int grid_dim = total_count >> 11;
  ls_dropout_act_bias_kernel<ActivationType::kRelu>
      <<<grid_dim + 1, 256, 0, stream>>>(
          total_count, ratio, out, vals, mask, bias,
          std::chrono::duration_cast<std::chrono::microseconds>(
              std::chrono::system_clock::now().time_since_epoch())
              .count(),
          dim);
}

/**
 * @brief fused bias, activation, and dropout at the end of first ffn
 *
 * @thread
 * gridDim.x = hidden_size / 8
 * blockDim.x = 8
 * blockDim.y = 1024 / 8 = 128
 *
 * @tparam act_type activation function, like kRelu, kGelu
 * @param total_count total elements
 * @param ratio drop ratio
 * @param out [batch_size, seq_len, hidden_size], float and __half
 * @param in [batch_size, seq_len, hidden_size], float and __half
 * @param mask [batch_size, seq_len, hidden_size], uint8 type
 * @param bias [hidden_size], ffn bias
 * @param seed seed to hiprand
 * @param hidden_size
 * @return void
 */
template <ActivationType act_type>
__global__ void ls_quant_dropout_act_bias_kernel(
    const int total_count, const float ratio, int8_t *qout, uint8_t *cmask_out,
    uint8_t *cmask_in, uint8_t *dropout_mask, const int8_t *qin,
    const float *bias, const float *cmax_out, const float *cmax_in,
    const int seed, const int hidden_size) {
  const float scale = 1.f / (1.f - ratio);
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 4 >= total_count) return;

  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, i, 0, &state);
  uint8_t m[4];

  float output_clip_max = cmax_out[0];
  float input_clip_max = cmax_in[0];

  int32_t *out4 = reinterpret_cast<int32_t *>(qout);
  const int32_t *qin4 = reinterpret_cast<const int32_t *>(qin);
  const float4 *bias4 = reinterpret_cast<const float4 *>(bias);
  uint32_t *dropout_mask4 = reinterpret_cast<uint32_t *>(dropout_mask);
  uint32_t *in_cmask4 = reinterpret_cast<uint32_t *>(cmask_in);
  float4 rand = hiprand_uniform4(&state);

  m[0] = (uint8_t)(rand.x > ratio);
  m[1] = (uint8_t)(rand.y > ratio);
  m[2] = (uint8_t)(rand.z > ratio);
  m[3] = (uint8_t)(rand.w > ratio);

  int bias_i = i % (hidden_size >> 2);
  uint32_t *m4 = reinterpret_cast<uint32_t *>(m);
  dropout_mask4[i] |= m4[0];
  int32_t qinput4 = qin4[i];
  int8_t *qinput = reinterpret_cast<int8_t *>(&qinput4);
  const float4 b4 = __ldg(&bias4[bias_i]);
  uint8_t in_cmask[4];
  int8_t out[4];

  out[0] = quantize(activation_kernel<act_type, float>(
                        dequantize(qinput[0], output_clip_max) + b4.x) *
                        scale * m[0],
                    input_clip_max, in_cmask[0], 2);
  out[1] = quantize(activation_kernel<act_type, float>(
                        dequantize(qinput[1], output_clip_max) + b4.y) *
                        scale * m[1],
                    input_clip_max, in_cmask[1], 2);
  out[2] = quantize(activation_kernel<act_type, float>(
                        dequantize(qinput[2], output_clip_max) + b4.z) *
                        scale * m[2],
                    input_clip_max, in_cmask[2], 2);
  out[3] = quantize(activation_kernel<act_type, float>(
                        dequantize(qinput[3], output_clip_max) + b4.w) *
                        scale * m[3],
                    input_clip_max, in_cmask[3], 2);

  in_cmask4[i] |= reinterpret_cast<uint32_t *>(in_cmask)[0];
  out4[i] = reinterpret_cast<int32_t *>(out)[0];
}

template <ActivationType act_type>
__global__ void ls_quant_dropout_act_bias_kernel(
    const int total_count, const float ratio, int8_t *qout, uint8_t *cmask_out,
    uint8_t *cmask_in, uint8_t *dropout_mask, const int8_t *qin,
    const __half *bias, const __half *cmax_out, const __half *cmax_in,
    const int seed, const int hidden_size) {
  const float scale = 1.f / (1.f - ratio);

  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 8 >= total_count) return;

  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, i, 0, &state);

  const int64_t *qin8 = reinterpret_cast<const int64_t *>(qin);
  int64_t *qout8 = reinterpret_cast<int64_t *>(qout);
  const float4 *bias4 = reinterpret_cast<const float4 *>(bias);
  uint64_t *dropout_mask8 = reinterpret_cast<uint64_t *>(dropout_mask);
  uint64_t *in_cmask8 = reinterpret_cast<uint64_t *>(cmask_in);

  float output_clip_max = __half2float(cmax_out[0]);
  float input_clip_max = __half2float(cmax_in[0]);

  uint8_t m[8];
  float4 rand = hiprand_uniform4(&state);
  m[0] = (uint8_t)(rand.x > ratio);
  m[1] = (uint8_t)(rand.y > ratio);
  m[2] = (uint8_t)(rand.z > ratio);
  m[3] = (uint8_t)(rand.w > ratio);
  rand = hiprand_uniform4(&state);
  m[4] = (uint8_t)(rand.x > ratio);
  m[5] = (uint8_t)(rand.y > ratio);
  m[6] = (uint8_t)(rand.z > ratio);
  m[7] = (uint8_t)(rand.w > ratio);
  uint64_t *m8 = reinterpret_cast<uint64_t *>(m);
  dropout_mask8[i] |= m8[0];

  int bias_i = i % (hidden_size >> 3);
  int64_t qinput8 = qin8[i];
  const float4 b4 = __ldg(&bias4[bias_i]);

  int8_t *qinput = reinterpret_cast<int8_t *>(&qinput8);
  int8_t out[8];
  const __half2 *b_half2 = reinterpret_cast<const __half2 *>(&b4);

  __half2 scale_mask[4];

  scale_mask[0] = __floats2half2_rn(scale * m[0], scale * m[1]);
  scale_mask[1] = __floats2half2_rn(scale * m[2], scale * m[3]);
  scale_mask[2] = __floats2half2_rn(scale * m[4], scale * m[5]);
  scale_mask[3] = __floats2half2_rn(scale * m[6], scale * m[7]);

  uint8_t in_cmask[8];

  __half2 temp;
#pragma unroll
  for (int j = 0; j < 4; j++) {
    temp.x = __float2half(dequantize(qinput[j * 2], output_clip_max));
    temp.y = __float2half(dequantize(qinput[j * 2 + 1], output_clip_max));

    temp =
        __hmul2(activation_kernel<act_type, __half2>(__hadd2(temp, b_half2[j])),
                scale_mask[j]);

    out[j * 2] =
        quantize(__half2float(temp.x), input_clip_max, in_cmask[j * 2], 2);
    out[j * 2 + 1] =
        quantize(__half2float(temp.y), input_clip_max, in_cmask[j * 2 + 1], 2);
  }

  in_cmask8[i] |= reinterpret_cast<uint64_t *>(in_cmask)[0];
  qout8[i] = reinterpret_cast<int64_t *>(out)[0];
}

template <>
void launch_ls_quant_dropout_act_bias<ActivationType::kGelu, float>(
    int8_t *qout, uint8_t *cmask_out, uint8_t *cmask_in, uint8_t *dropout_mask,
    const int8_t *qinput, const float *bias, const float *cmax_out,
    const float *cmax_in, int total_count, int dim, float ratio,
    hipStream_t stream) {
  int grid_dim = total_count >> 10;
  ls_quant_dropout_act_bias_kernel<ActivationType::kGelu>
      <<<grid_dim + 1, 256, 0, stream>>>(
          total_count, ratio, qout, cmask_out, cmask_in, dropout_mask, qinput,
          bias, cmax_out, cmax_in,
          std::chrono::duration_cast<std::chrono::microseconds>(
              std::chrono::system_clock::now().time_since_epoch())
              .count(),
          dim);
}

template <>
void launch_ls_quant_dropout_act_bias<ActivationType::kGelu, __half>(
    int8_t *qout, uint8_t *cmask_out, uint8_t *cmask_in, uint8_t *dropout_mask,
    const int8_t *qinput, const __half *bias, const __half *cmax_out,
    const __half *cmax_in, int total_count, int dim, float ratio,
    hipStream_t stream) {
  int grid_dim = total_count >> 11;
  ls_quant_dropout_act_bias_kernel<ActivationType::kGelu>
      <<<grid_dim + 1, 256, 0, stream>>>(
          total_count, ratio, qout, cmask_out, cmask_in, dropout_mask, qinput,
          bias, cmax_out, cmax_in,
          std::chrono::duration_cast<std::chrono::microseconds>(
              std::chrono::system_clock::now().time_since_epoch())
              .count(),
          dim);
}

template <>
void launch_ls_quant_dropout_act_bias<ActivationType::kRelu, float>(
    int8_t *qout, uint8_t *cmask_out, uint8_t *cmask_in, uint8_t *dropout_mask,
    const int8_t *qinput, const float *bias, const float *cmax_out,
    const float *cmax_in, int total_count, int dim, float ratio,
    hipStream_t stream) {
  int grid_dim = total_count >> 10;
  ls_quant_dropout_act_bias_kernel<ActivationType::kRelu>
      <<<grid_dim + 1, 256, 0, stream>>>(
          total_count, ratio, qout, cmask_out, cmask_in, dropout_mask, qinput,
          bias, cmax_out, cmax_in,
          std::chrono::duration_cast<std::chrono::microseconds>(
              std::chrono::system_clock::now().time_since_epoch())
              .count(),
          dim);
}

template <>
void launch_ls_quant_dropout_act_bias<ActivationType::kRelu, __half>(
    int8_t *qout, uint8_t *cmask_out, uint8_t *cmask_in, uint8_t *dropout_mask,
    const int8_t *qinput, const __half *bias, const __half *cmax_out,
    const __half *cmax_in, int total_count, int dim, float ratio,
    hipStream_t stream) {
  int grid_dim = total_count >> 11;
  ls_quant_dropout_act_bias_kernel<ActivationType::kRelu>
      <<<grid_dim + 1, 256, 0, stream>>>(
          total_count, ratio, qout, cmask_out, cmask_in, dropout_mask, qinput,
          bias, cmax_out, cmax_in,
          std::chrono::duration_cast<std::chrono::microseconds>(
              std::chrono::system_clock::now().time_since_epoch())
              .count(),
          dim);
}

/**
 * @brief fused bias, activation, and dropout at the end of first ffn
 *
 * @thread
 * gridDim.x = hidden_size / 8
 * blockDim.x = 8
 * blockDim.y = 1024 / 8 = 128
 *
 * @tparam act_type activation function, like kRelu, kGelu
 * @param total_count total elements
 * @param ratio drop ratio
 * @param out [batch_size, seq_len, hidden_size], float and __half
 * @param in [batch_size, seq_len, hidden_size], float and __half
 * @param mask [batch_size, seq_len, hidden_size], uint8 type
 * @param bias [hidden_size], ffn bias
 * @param seed seed to hiprand
 * @param hidden_size
 * @return void
 */
template <ActivationType act_type>
__global__ void ls_fakequant_dropout_act_bias_kernel(
    const int total_count, const float ratio, float *qout, uint8_t *cmask_out,
    uint8_t *cmask_in, uint8_t *dropout_mask, const int8_t *qin,
    const float *bias, const float *cmax_out, const float *cmax_in,
    const int seed, const int hidden_size, bool in_col32, bool symmetry) {
  const float scale = 1.f / (1.f - ratio);
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 4 >= total_count) return;

  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, i, 0, &state);
  uint8_t m[4];

  float output_clip_max = cmax_out[0];
  float input_clip_max = cmax_in[0];

  float4 *out4 = reinterpret_cast<float4 *>(qout);
  const int32_t *qin4 = reinterpret_cast<const int32_t *>(qin);
  const float4 *bias4 = reinterpret_cast<const float4 *>(bias);
  uint32_t *dropout_mask4 = reinterpret_cast<uint32_t *>(dropout_mask);
  uint32_t *in_cmask4 = reinterpret_cast<uint32_t *>(cmask_in);
  float4 rand = hiprand_uniform4(&state);

  m[0] = (uint8_t)(rand.x > ratio);
  m[1] = (uint8_t)(rand.y > ratio);
  m[2] = (uint8_t)(rand.z > ratio);
  m[3] = (uint8_t)(rand.w > ratio);

  int bias_i = i % (hidden_size >> 2);
  uint32_t *m4 = reinterpret_cast<uint32_t *>(m);
  dropout_mask4[i] |= m4[0];

  int input_index;
  if (in_col32) {
    int batch_tokens = total_count / hidden_size;
    int row_id = (i * 4) / hidden_size;
    int col_id = (i * 4) % hidden_size;
    input_index =
        row_major2flat_col32(row_id, col_id, batch_tokens, hidden_size) / 4;
  } else {
    input_index = i;
  }

  int32_t qinput4 = qin4[input_index];
  int8_t *qinput = reinterpret_cast<int8_t *>(&qinput4);
  const float4 b4 = __ldg(&bias4[bias_i]);
  uint8_t in_cmask[4];
  float4 out;

  out.x = fake_quantize(activation_kernel<act_type, float>(
                            dequantize(qinput[0], output_clip_max) + b4.x) *
                            scale * m[0],
                        input_clip_max, in_cmask[0], 2, symmetry);
  out.y = fake_quantize(activation_kernel<act_type, float>(
                            dequantize(qinput[1], output_clip_max) + b4.y) *
                            scale * m[1],
                        input_clip_max, in_cmask[1], 2, symmetry);
  out.z = fake_quantize(activation_kernel<act_type, float>(
                            dequantize(qinput[2], output_clip_max) + b4.z) *
                            scale * m[2],
                        input_clip_max, in_cmask[2], 2, symmetry);
  out.w = fake_quantize(activation_kernel<act_type, float>(
                            dequantize(qinput[3], output_clip_max) + b4.w) *
                            scale * m[3],
                        input_clip_max, in_cmask[3], 2, symmetry);

  in_cmask4[i] |= reinterpret_cast<uint32_t *>(in_cmask)[0];
  out4[i] = out;
}

template <ActivationType act_type>
__global__ void ls_fakequant_dropout_act_bias_kernel(
    const int total_count, const float ratio, __half *qout, uint8_t *cmask_out,
    uint8_t *cmask_in, uint8_t *dropout_mask, const int8_t *qin,
    const __half *bias, const __half *cmax_out, const __half *cmax_in,
    const int seed, const int hidden_size, bool in_col32, bool symmetry) {
  const float scale = 1.f / (1.f - ratio);

  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 8 >= total_count) return;

  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, i, 0, &state);

  const int64_t *qin8 = reinterpret_cast<const int64_t *>(qin);
  float4 *qout8 = reinterpret_cast<float4 *>(qout);
  const float4 *bias4 = reinterpret_cast<const float4 *>(bias);
  uint64_t *dropout_mask8 = reinterpret_cast<uint64_t *>(dropout_mask);
  uint64_t *in_cmask8 = reinterpret_cast<uint64_t *>(cmask_in);

  float output_clip_max = __half2float(cmax_out[0]);
  float input_clip_max = __half2float(cmax_in[0]);

  uint8_t m[8];
  float4 rand = hiprand_uniform4(&state);
  m[0] = (uint8_t)(rand.x > ratio);
  m[1] = (uint8_t)(rand.y > ratio);
  m[2] = (uint8_t)(rand.z > ratio);
  m[3] = (uint8_t)(rand.w > ratio);
  rand = hiprand_uniform4(&state);
  m[4] = (uint8_t)(rand.x > ratio);
  m[5] = (uint8_t)(rand.y > ratio);
  m[6] = (uint8_t)(rand.z > ratio);
  m[7] = (uint8_t)(rand.w > ratio);
  uint64_t *m8 = reinterpret_cast<uint64_t *>(m);
  dropout_mask8[i] |= m8[0];

  int bias_i = i % (hidden_size >> 3);
  int input_index;
  if (in_col32) {
    int batch_tokens = total_count / hidden_size;
    int row_id = (i * 8) / hidden_size;
    int col_id = (i * 8) % hidden_size;
    input_index =
        row_major2flat_col32(row_id, col_id, batch_tokens, hidden_size) / 8;
  } else {
    input_index = i;
  }
  int64_t qinput8 = qin8[input_index];
  const float4 b4 = __ldg(&bias4[bias_i]);

  int8_t *qinput = reinterpret_cast<int8_t *>(&qinput8);
  float4 out8;
  __half2 *out = reinterpret_cast<__half2 *>(&out8);
  const __half2 *b_half2 = reinterpret_cast<const __half2 *>(&b4);

  __half2 scale_mask[4];

  scale_mask[0] = __floats2half2_rn(scale * m[0], scale * m[1]);
  scale_mask[1] = __floats2half2_rn(scale * m[2], scale * m[3]);
  scale_mask[2] = __floats2half2_rn(scale * m[4], scale * m[5]);
  scale_mask[3] = __floats2half2_rn(scale * m[6], scale * m[7]);

  uint8_t in_cmask[8];

  __half2 temp;
#pragma unroll
  for (int j = 0; j < 4; j++) {
    temp.x = __float2half(dequantize(qinput[j * 2], output_clip_max));
    temp.y = __float2half(dequantize(qinput[j * 2 + 1], output_clip_max));

    temp =
        __hmul2(activation_kernel<act_type, __half2>(__hadd2(temp, b_half2[j])),
                scale_mask[j]);

    out[j].x = __float2half(fake_quantize(__half2float(temp.x), input_clip_max,
                                          in_cmask[j * 2], 2, symmetry));
    out[j].y = __float2half(fake_quantize(__half2float(temp.y), input_clip_max,
                                          in_cmask[j * 2 + 1], 2, symmetry));
  }

  in_cmask8[i] |= reinterpret_cast<uint64_t *>(in_cmask)[0];
  qout8[i] = out8;
}

template <>
void launch_ls_fakequant_dropout_act_bias<ActivationType::kGelu, float>(
    float *out, uint8_t *cmask_out, uint8_t *cmask_in, uint8_t *dropout_mask,
    const int8_t *qinput, const float *bias, const float *cmax_out,
    const float *cmax_in, int total_count, int dim, float ratio,
    hipStream_t stream, bool in_col32, bool symmetry) {
  int grid_dim = total_count >> 10;
  ls_fakequant_dropout_act_bias_kernel<ActivationType::kGelu>
      <<<grid_dim + 1, 256, 0, stream>>>(
          total_count, ratio, out, cmask_out, cmask_in, dropout_mask, qinput,
          bias, cmax_out, cmax_in,
          std::chrono::duration_cast<std::chrono::microseconds>(
              std::chrono::system_clock::now().time_since_epoch())
              .count(),
          dim, in_col32, symmetry);
}

template <>
void launch_ls_fakequant_dropout_act_bias<ActivationType::kGelu, __half>(
    __half *out, uint8_t *cmask_out, uint8_t *cmask_in, uint8_t *dropout_mask,
    const int8_t *qinput, const __half *bias, const __half *cmax_out,
    const __half *cmax_in, int total_count, int dim, float ratio,
    hipStream_t stream, bool in_col32, bool symmetry) {
  int grid_dim = total_count >> 11;
  ls_fakequant_dropout_act_bias_kernel<ActivationType::kGelu>
      <<<grid_dim + 1, 256, 0, stream>>>(
          total_count, ratio, out, cmask_out, cmask_in, dropout_mask, qinput,
          bias, cmax_out, cmax_in,
          std::chrono::duration_cast<std::chrono::microseconds>(
              std::chrono::system_clock::now().time_since_epoch())
              .count(),
          dim, in_col32, symmetry);
}

template <>
void launch_ls_fakequant_dropout_act_bias<ActivationType::kRelu, float>(
    float *out, uint8_t *cmask_out, uint8_t *cmask_in, uint8_t *dropout_mask,
    const int8_t *qinput, const float *bias, const float *cmax_out,
    const float *cmax_in, int total_count, int dim, float ratio,
    hipStream_t stream, bool in_col32, bool symmetry) {
  int grid_dim = total_count >> 10;
  ls_fakequant_dropout_act_bias_kernel<ActivationType::kRelu>
      <<<grid_dim + 1, 256, 0, stream>>>(
          total_count, ratio, out, cmask_out, cmask_in, dropout_mask, qinput,
          bias, cmax_out, cmax_in,
          std::chrono::duration_cast<std::chrono::microseconds>(
              std::chrono::system_clock::now().time_since_epoch())
              .count(),
          dim, in_col32, symmetry);
}

template <>
void launch_ls_fakequant_dropout_act_bias<ActivationType::kRelu, __half>(
    __half *out, uint8_t *cmask_out, uint8_t *cmask_in, uint8_t *dropout_mask,
    const int8_t *qinput, const __half *bias, const __half *cmax_out,
    const __half *cmax_in, int total_count, int dim, float ratio,
    hipStream_t stream, bool in_col32, bool symmetry) {
  int grid_dim = total_count >> 11;
  ls_fakequant_dropout_act_bias_kernel<ActivationType::kRelu>
      <<<grid_dim + 1, 256, 0, stream>>>(
          total_count, ratio, out, cmask_out, cmask_in, dropout_mask, qinput,
          bias, cmax_out, cmax_in,
          std::chrono::duration_cast<std::chrono::microseconds>(
              std::chrono::system_clock::now().time_since_epoch())
              .count(),
          dim, in_col32, symmetry);
}

/**
 * @brief fused bias, activation, and dropout backward
 *
 * @thread
 * gridDim.x = total_count / 1024
 * blockDim.x = 1024
 *
 * @tparam act_type kRelu
 * @param row_size batch_size * seq_len
 * @param ratio dropout ratio
 * @param in_grad [batch_size, seq_len, hidden_size], input grad
 * @param bias_grad [hidden_size], bias grad
 * @param out_grad [batch_size, seq_len, hidden_size], output grad
 * @param mask [batch_size, seq_len, hidden_size], dropout mask
 * @param hidden_size
 * @return void
 */
template <ActivationType act_type, typename T>
__global__ void ls_dropout_act_bias_bwd_kernel(
    const int row_size, const float ratio, T *in_grad,
    T *__restrict__ bias_grad, const T *__restrict__ input,
    const T *__restrict__ bias, const T *out_grad,
    const uint8_t *__restrict__ mask, const int hidden_size) {
  const float scale = 1.f / (1.f - ratio);
  __shared__ float tile[WARP_SIZE][WARP_SIZE + 1];

  cg::thread_block b = cg::this_thread_block();
  cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

  int col_idx = flat_2dim(blockIdx.x, threadIdx.x, WARP_SIZE);

  int stride = hidden_size * WARP_SIZE;
  float local_sum = 0;

  int idx = flat_2dim(threadIdx.y, col_idx, hidden_size);
  if (col_idx < hidden_size) {
    for (int r = threadIdx.y; r < row_size; r += WARP_SIZE) {
      float val = out_grad[idx];
      float in = input[idx];
      float b = bias[idx % hidden_size];
      val = activation_bwd_kernel<act_type, float>(
          val * scale * static_cast<float>(mask[idx]), in + b);
      local_sum += val;
      in_grad[idx] = val;
      idx += stride;
    }
  }

  tile[threadIdx.x][threadIdx.y] = local_sum;
  __syncthreads();
  float sum = tile[threadIdx.y][threadIdx.x];
  __syncthreads();

  for (int i = 1; i < WARP_SIZE; i <<= 1) sum += g.shfl_down(sum, i);

  if (threadIdx.x == 0) tile[0][threadIdx.y] = sum;
  __syncthreads();

  if (threadIdx.y == 0) {
    int pos = flat_2dim(blockIdx.x, threadIdx.x, WARP_SIZE);
    bias_grad[pos] = tile[0][threadIdx.x];
  }
}

// @brief fused bias, activation, and dropout backward
// It is deprecated for precision reason. Keep it for future optimization.
//
// template <ActivationType act_type>
// __global__ void ls_dropout_act_bias_bwd_kernel(
//     const int row_size, const float ratio, __half * in_grad,
//     __half *__restrict__ bias_grad, const __half *__restrict__ input, const
//     __half *__restrict__ bias, const __half * out_grad, const uint8_t
//     *__restrict__ mask, const int hidden_size) {
//   const __half2 scale = __float2half2_rn(1.f / (1.f - ratio));
//   __shared__ __half2 tile[WARP_SIZE][WARP_SIZE + 1];

//   cg::thread_block b = cg::this_thread_block();
//   cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

//   __half2 *in_grad2 = reinterpret_cast<__half2 *>(in_grad);
//   __half2 *bias_grad2 = reinterpret_cast<__half2 *>(bias_grad);
//   const __half2 *out_grad2 = reinterpret_cast<const __half2 *>(out_grad);
//   const __half2 *input2 = reinterpret_cast<const __half2 *>(input);
//   const __half2 *bias2 = reinterpret_cast<const __half2 *>(bias);

//   int col_idx = flat_2dim(blockIdx.x, threadIdx.x, WARP_SIZE);

//   int stride = hidden_size * WARP_SIZE;
//   __half2 local_sum = __float2half2_rn(0.f);

//   int idx = flat_2dim(threadIdx.y, col_idx, hidden_size);
//   if (col_idx < hidden_size) {
//     for (int r = threadIdx.y; r < row_size; r += WARP_SIZE) {
//       __half2 val = out_grad2[idx];
//       __half2 in2 = input2[idx];
//       __half2 b2 = bias2[idx % hidden_size ];
//       __half2 m2 = __floats2half2_rn(mask[2 * idx], mask[2 * idx + 1]);
//       val = activation_bwd_kernel<ActivationType::kRelu, __half2>(val * scale
//       *
//                                                                   m2,
//                                                                   in2+b2);
//       local_sum += val;
//       in_grad2[idx] = val;
//       idx += stride;
//     }
//   }

//   tile[threadIdx.x][threadIdx.y] = local_sum;
//   __syncthreads();
//   __half2 sum = tile[threadIdx.y][threadIdx.x];
//   __syncthreads();

//   for (int i = 1; i < WARP_SIZE; i <<= 1) sum += g.shfl_down(sum, i);

//   if (threadIdx.x == 0) tile[0][threadIdx.y] = sum;
//   __syncthreads();

//   if (threadIdx.y == 0) {
//     int pos = flat_2dim(blockIdx.x, threadIdx.x, WARP_SIZE);
//     bias_grad2[pos] = tile[0][threadIdx.x];
//   }
// }

template <ActivationType act_type, typename T>
void launch_ls_dropout_act_bias_bwd(T *in_grad, T *bias_grad, const T *input,
                                    const T *bias, const T *out_grad,
                                    const uint8_t *mask, int row_size, int dim,
                                    float ratio, hipStream_t stream) {
  dim3 grid_dim((dim - 1) / WARP_SIZE + 1);
  dim3 block_dim(WARP_SIZE, WARP_SIZE);
  ls_dropout_act_bias_bwd_kernel<act_type><<<grid_dim, block_dim, 0, stream>>>(
      row_size, ratio, in_grad, bias_grad, input, bias, out_grad, mask, dim);
}

// template <>
// void launch_ls_dropout_act_bias_bwd<ActivationType::kRelu, __half>(
//     __half *in_grad, __half *bias_grad,const __half *input, const __half
//     *bias, const __half *out_grad, const uint8_t *mask, int row_size, int
//     dim, float ratio, hipStream_t stream) {
//   dim >>= 1;
//   dim3 grid_dim((dim - 1) / WARP_SIZE + 1);
//   dim3 block_dim(WARP_SIZE, WARP_SIZE);
//   ls_dropout_act_bias_bwd_kernel<ActivationType::kRelu>
//       <<<grid_dim, block_dim, 0, stream>>>(row_size, ratio, in_grad,
//       bias_grad,
//                                            input, bias,out_grad, mask, dim);
// }

template void launch_ls_dropout_act_bias_bwd<ActivationType::kRelu, float>(
    float *in_grad, float *bias_grad, const float *input, const float *bias,
    const float *out_grad, const uint8_t *mask, int row_size, int dim,
    float ratio, hipStream_t stream);

template void launch_ls_dropout_act_bias_bwd<ActivationType::kRelu, __half>(
    __half *in_grad, __half *bias_grad, const __half *input, const __half *bias,
    const __half *out_grad, const uint8_t *mask, int row_size, int dim,
    float ratio, hipStream_t stream);

template void launch_ls_dropout_act_bias_bwd<ActivationType::kGelu, float>(
    float *in_grad, float *bias_grad, const float *input, const float *bias,
    const float *out_grad, const uint8_t *mask, int row_size, int dim,
    float ratio, hipStream_t stream);

template void launch_ls_dropout_act_bias_bwd<ActivationType::kGelu, __half>(
    __half *in_grad, __half *bias_grad, const __half *input, const __half *bias,
    const __half *out_grad, const uint8_t *mask, int row_size, int dim,
    float ratio, hipStream_t stream);

/**
 * @brief fused bias, activation, and dropout backward
 *
 * @thread
 * gridDim.x = total_count / 1024
 * blockDim.x = 1024
 *
 * @tparam act_type kRelu
 * @param row_size batch_size * seq_len
 * @param ratio dropout ratio
 * @param in_grad [batch_size, seq_len, hidden_size], input grad
 * @param bias_grad [hidden_size], bias grad
 * @param out_grad [batch_size, seq_len, hidden_size], output grad
 * @param mask [batch_size, seq_len, hidden_size], dropout mask
 * @param hidden_size
 * @return void
 */
template <ActivationType act_type, typename T>
__global__ void ls_quant_dropout_act_bias_bwd_kernel(
    T *in_grad, T *bias_grad, T *cmax_in_grad, T *cmax_out_grad,
    const int8_t *input, const T *cmax_in, const uint8_t *cmask_in,
    const uint8_t *cmask_out, const T *bias, const T *out_grad,
    const uint8_t *dropout_mask, int row_size, float ratio, int hidden_size,
    bool in_col32) {
  const float scale = 1.f / (1.f - ratio);
  __shared__ float tile[WARP_SIZE][WARP_SIZE + 1];

  cg::thread_block b = cg::this_thread_block();
  cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

  int col_idx = flat_2dim(blockIdx.x, threadIdx.x, WARP_SIZE);

  int stride = hidden_size * WARP_SIZE;
  float thread_grad_bias = 0;

  float cmax_in_val = cmax_in[0];

  int idx = flat_2dim(threadIdx.y, col_idx, hidden_size);

  // float thread_cmax_out_grad = 0;
  float thread_cmax_in_grad = 0;
  float thread_in_grad = 0;
  float temp_cmax_in_grad = 0;
  // float temp_cmax_out_grad = 0;

  int input_index;

  if (col_idx < hidden_size) {
    for (int r = threadIdx.y; r < row_size; r += WARP_SIZE) {
      float val = out_grad[idx];
      // clip_bwd(thread_in_grad, temp_cmax_out_grad, float{out_grad[idx]},
      //          cmask_out[idx], 2);
      // thread_cmax_out_grad += temp_cmax_out_grad;
      if (in_col32) {
        int row_id = idx / hidden_size;
        int col_id = idx % hidden_size;
        input_index =
            row_major2flat_col32(row_id, col_id, row_size, hidden_size);
      } else {
        input_index = idx;
      }

      float in = dequantize(input[input_index], cmax_in_val);
      float b = bias[idx % hidden_size];
      uint8_t mask = dropout_mask[idx];
      thread_in_grad = activation_bwd_kernel<act_type, float>(
          val * scale * static_cast<float>(mask & 1), in + b);
      thread_grad_bias += thread_in_grad;

      clip_bwd(thread_in_grad, temp_cmax_in_grad, thread_in_grad, mask, 6);
      in_grad[idx] = thread_in_grad;
      thread_cmax_in_grad += temp_cmax_in_grad;
      idx += stride;
    }
  }
  __shared__ float block_cmax_in_grad;
  // __shared__ float block_cmax_out_grad;
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    block_cmax_in_grad = 0;
    // block_cmax_out_grad = 0;
  }

  tile[threadIdx.x][threadIdx.y] = thread_grad_bias;
  __syncthreads();
  // if (thread_cmax_out_grad != 0) {
  //   atomicAdd(&block_cmax_out_grad, thread_cmax_out_grad);
  // }
  if (thread_cmax_in_grad != 0) {
    atomicAdd(&block_cmax_in_grad, thread_cmax_in_grad);
  }

  float sum = tile[threadIdx.y][threadIdx.x];

  __syncthreads();

  if (threadIdx.x == 0 && threadIdx.y == 0) {
    if (block_cmax_in_grad != 0) {
      atomicAdd(&cmax_in_grad[0], block_cmax_in_grad);
    }
    // if (block_cmax_out_grad != 0) {
    //   atomicAdd(&cmax_out_grad[0], block_cmax_out_grad);
    // }
  }
  for (int i = 1; i < WARP_SIZE; i <<= 1) sum += g.shfl_down(sum, i);

  if (threadIdx.x == 0) tile[0][threadIdx.y] = sum;
  __syncthreads();

  if (threadIdx.y == 0) {
    int pos = flat_2dim(blockIdx.x, threadIdx.x, WARP_SIZE);
    bias_grad[pos] = tile[0][threadIdx.x];
  }
}

template <ActivationType act_type, typename T>
void launch_ls_quant_dropout_act_bias_bwd(
    T *in_grad, T *bias_grad, T *cmax_in_grad, T *cmax_out_grad,
    const int8_t *input, const T *cmax_in, const uint8_t *cmask_in,
    const uint8_t *cmask_out, const T *bias, const T *out_grad,
    const uint8_t *dropout_mask, int row_size, int dim, float ratio,
    hipStream_t stream, bool in_col32) {
  zero_grad<<<1, 1>>>(cmax_in_grad);
  zero_grad<<<1, 1>>>(cmax_out_grad);
  dim3 grid_dim((dim - 1) / WARP_SIZE + 1);
  dim3 block_dim(WARP_SIZE, WARP_SIZE);
  ls_quant_dropout_act_bias_bwd_kernel<act_type>
      <<<grid_dim, block_dim, 0, stream>>>(
          in_grad, bias_grad, cmax_in_grad, cmax_out_grad, input, cmax_in,
          cmask_in, cmask_out, bias, out_grad, dropout_mask, row_size, ratio,
          dim, in_col32);
}

template void
launch_ls_quant_dropout_act_bias_bwd<ActivationType::kRelu, float>(
    float *in_grad, float *bias_grad, float *cmax_in_grad, float *cmax_out_grad,
    const int8_t *input, const float *cmax_in, const uint8_t *cmask_in,
    const uint8_t *cmask_out, const float *bias, const float *out_grad,
    const uint8_t *dropout_mask, int row_size, int dim, float ratio,
    hipStream_t stream, bool in_col32);

template void
launch_ls_quant_dropout_act_bias_bwd<ActivationType::kRelu, __half>(
    __half *in_grad, __half *bias_grad, __half *cmax_in_grad,
    __half *cmax_out_grad, const int8_t *input, const __half *cmax_in,
    const uint8_t *cmask_in, const uint8_t *cmask_out, const __half *bias,
    const __half *out_grad, const uint8_t *dropout_mask, int row_size, int dim,
    float ratio, hipStream_t stream, bool in_col32);

template void
launch_ls_quant_dropout_act_bias_bwd<ActivationType::kGelu, float>(
    float *in_grad, float *bias_grad, float *cmax_in_grad, float *cmax_out_grad,
    const int8_t *input, const float *cmax_in, const uint8_t *cmask_in,
    const uint8_t *cmask_out, const float *bias, const float *out_grad,
    const uint8_t *dropout_mask, int row_size, int dim, float ratio,
    hipStream_t stream, bool in_col32);

template void
launch_ls_quant_dropout_act_bias_bwd<ActivationType::kGelu, __half>(
    __half *in_grad, __half *bias_grad, __half *cmax_in_grad,
    __half *cmax_out_grad, const int8_t *input, const __half *cmax_in,
    const uint8_t *cmask_in, const uint8_t *cmask_out, const __half *bias,
    const __half *out_grad, const uint8_t *dropout_mask, int row_size, int dim,
    float ratio, hipStream_t stream, bool in_col32);

/**
 * @brief fused bias, activation, and dropout backward, with float input
 *
 * @thread
 * gridDim.x = total_count / 1024
 * blockDim.x = 1024
 *
 * @tparam act_type kRelu
 * @param row_size batch_size * seq_len
 * @param ratio dropout ratio
 * @param in_grad [batch_size, seq_len, hidden_size], input grad
 * @param bias_grad [hidden_size], bias grad
 * @param out_grad [batch_size, seq_len, hidden_size], output grad
 * @param mask [batch_size, seq_len, hidden_size], dropout mask
 * @param hidden_size
 * @return void
 */
template <ActivationType act_type, typename T>
__global__ void ls_quant_dropout_act_bias_bwd_kernel(
    T *in_grad, T *bias_grad, T *cmax_in_grad, T *cmax_out_grad, const T *input,
    const T *cmax_in, const uint8_t *cmask_in, const uint8_t *cmask_out,
    const T *bias, const T *out_grad, const uint8_t *dropout_mask, int row_size,
    float ratio, int hidden_size) {
  const float scale = 1.f / (1.f - ratio);
  __shared__ float tile[WARP_SIZE][WARP_SIZE + 1];

  cg::thread_block b = cg::this_thread_block();
  cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

  int col_idx = flat_2dim(blockIdx.x, threadIdx.x, WARP_SIZE);

  int stride = hidden_size * WARP_SIZE;
  float thread_grad_bias = 0;

  float cmax_in_val = cmax_in[0];

  int idx = flat_2dim(threadIdx.y, col_idx, hidden_size);

  float thread_cmax_out_grad = 0;
  float thread_cmax_in_grad = 0;
  float thread_in_grad = 0;
  float temp_cmax_in_grad = 0;
  float temp_cmax_out_grad = 0;
  if (col_idx < hidden_size) {
    for (int r = threadIdx.y; r < row_size; r += WARP_SIZE) {
      // float val = out_grad[idx];
      clip_bwd(thread_in_grad, temp_cmax_out_grad, float{out_grad[idx]},
               cmask_out[idx], 2);
      thread_cmax_out_grad += temp_cmax_out_grad;

      float in = input[idx];
      float b = bias[idx % hidden_size];
      thread_in_grad = activation_bwd_kernel<act_type, float>(
          thread_in_grad * scale * static_cast<float>(dropout_mask[idx] & 1),
          in + b);
      thread_grad_bias += thread_in_grad;

      clip_bwd(thread_in_grad, temp_cmax_in_grad, thread_in_grad, cmask_in[idx],
               6);
      in_grad[idx] = thread_in_grad;
      thread_cmax_in_grad += temp_cmax_in_grad;
      idx += stride;
    }
  }
  __shared__ float block_cmax_in_grad, block_cmax_out_grad;
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    block_cmax_in_grad = 0;
    block_cmax_out_grad = 0;
  }
  __syncthreads();

  if (thread_cmax_out_grad != 0) {
    atomicAdd(&block_cmax_out_grad, thread_cmax_out_grad);
  }
  if (thread_cmax_in_grad != 0) {
    atomicAdd(&block_cmax_in_grad, thread_cmax_in_grad);
  }

  tile[threadIdx.x][threadIdx.y] = thread_grad_bias;
  __syncthreads();
  float sum = tile[threadIdx.y][threadIdx.x];

  if (threadIdx.x == 0 && threadIdx.y == 0) {
    if (block_cmax_in_grad != 0) {
      atomicAdd(&cmax_in_grad[0], block_cmax_in_grad);
    }
    if (block_cmax_out_grad != 0) {
      atomicAdd(&cmax_out_grad[0], block_cmax_out_grad);
    }
  }

  __syncthreads();

  for (int i = 1; i < WARP_SIZE; i <<= 1) sum += g.shfl_down(sum, i);

  if (threadIdx.x == 0) tile[0][threadIdx.y] = sum;
  __syncthreads();

  if (threadIdx.y == 0) {
    int pos = flat_2dim(blockIdx.x, threadIdx.x, WARP_SIZE);
    bias_grad[pos] = tile[0][threadIdx.x];
  }
}

template <ActivationType act_type, typename T>
void launch_ls_quant_dropout_act_bias_bwd(
    T *in_grad, T *bias_grad, T *cmax_in_grad, T *cmax_out_grad, const T *input,
    const T *cmax_in, const uint8_t *cmask_in, const uint8_t *cmask_out,
    const T *bias, const T *out_grad, const uint8_t *dropout_mask, int row_size,
    int dim, float ratio, hipStream_t stream) {
  zero_grad<<<1, 1>>>(cmax_in_grad);
  zero_grad<<<1, 1>>>(cmax_out_grad);
  dim3 grid_dim((dim - 1) / WARP_SIZE + 1);
  dim3 block_dim(WARP_SIZE, WARP_SIZE);
  ls_quant_dropout_act_bias_bwd_kernel<act_type>
      <<<grid_dim, block_dim, 0, stream>>>(in_grad, bias_grad, cmax_in_grad,
                                           cmax_out_grad, input, cmax_in,
                                           cmask_in, cmask_out, bias, out_grad,
                                           dropout_mask, row_size, ratio, dim);
}

template void
launch_ls_quant_dropout_act_bias_bwd<ActivationType::kRelu, float>(
    float *in_grad, float *bias_grad, float *cmax_in_grad, float *cmax_out_grad,
    const float *input, const float *cmax_in, const uint8_t *cmask_in,
    const uint8_t *cmask_out, const float *bias, const float *out_grad,
    const uint8_t *dropout_mask, int row_size, int dim, float ratio,
    hipStream_t stream);

template void
launch_ls_quant_dropout_act_bias_bwd<ActivationType::kRelu, __half>(
    __half *in_grad, __half *bias_grad, __half *cmax_in_grad,
    __half *cmax_out_grad, const __half *input, const __half *cmax_in,
    const uint8_t *cmask_in, const uint8_t *cmask_out, const __half *bias,
    const __half *out_grad, const uint8_t *dropout_mask, int row_size, int dim,
    float ratio, hipStream_t stream);

template void
launch_ls_quant_dropout_act_bias_bwd<ActivationType::kGelu, float>(
    float *in_grad, float *bias_grad, float *cmax_in_grad, float *cmax_out_grad,
    const float *input, const float *cmax_in, const uint8_t *cmask_in,
    const uint8_t *cmask_out, const float *bias, const float *out_grad,
    const uint8_t *dropout_mask, int row_size, int dim, float ratio,
    hipStream_t stream);

template void
launch_ls_quant_dropout_act_bias_bwd<ActivationType::kGelu, __half>(
    __half *in_grad, __half *bias_grad, __half *cmax_in_grad,
    __half *cmax_out_grad, const __half *input, const __half *cmax_in,
    const uint8_t *cmask_in, const uint8_t *cmask_out, const __half *bias,
    const __half *out_grad, const uint8_t *dropout_mask, int row_size, int dim,
    float ratio, hipStream_t stream);

/**
 * @brief fused bias, dropout, and residual at the end of Attention and FFN,
 * store dropped position in mask, it's not in-place
 *
 * @thread
 * gridDim.x = total_count / 1024
 * blockDim.x = 1024
 *
 * @param total_count total elements
 * @param ratio drop ratio
 * @param out [batch_size, seq_len, hidden_size], float and __half
 * @param in [batch_size, seq_len, hidden_size], float and __half
 * @param mask [batch_size, seq_len, hidden_size], uint8 type
 * @param bias [hidden_size], ffn bias
 * @param residual [batch_size, seq_len, hidden_size], float and __half
 * @param seed seed to hiprand
 * @param hidden_size hidden size
 * @return void
 */
__global__ void ls_quant_dropout_res_bias_kernel(
    const int total_count, const float ratio, float *out, uint8_t *mask,
    const int8_t *qin, const float *cmax, const float *bias,
    const float *residual, const int seed, const int hidden_size,
    bool in_col32) {
  const float scale = 1.f / (1.f - ratio);
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 4 >= total_count) return;

  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, i, 0, &state);
  uint8_t m[4];

  float4 *out4 = reinterpret_cast<float4 *>(out);
  const int32_t *qdata4 = reinterpret_cast<const int32_t *>(qin);
  const float4 *residual4 = reinterpret_cast<const float4 *>(residual);
  const float4 *bias4 = reinterpret_cast<const float4 *>(bias);
  uint32_t *mask4 = reinterpret_cast<uint32_t *>(mask);
  float4 rand = hiprand_uniform4(&state);

  m[0] = static_cast<uint8_t>(rand.x > ratio);
  m[1] = static_cast<uint8_t>(rand.y > ratio);
  m[2] = static_cast<uint8_t>(rand.z > ratio);
  m[3] = static_cast<uint8_t>(rand.w > ratio);

  float cmax_val = cmax[0];
  int bias_i = i % (hidden_size >> 2);
  uint32_t *m4 = reinterpret_cast<uint32_t *>(m);
  mask4[i] |= m4[0];

  int input_index;
  if (in_col32) {
    int batch_tokens = total_count / hidden_size;
    int row_id = (i * 4) / hidden_size;
    int col_id = (i * 4) % hidden_size;
    input_index =
        row_major2flat_col32(row_id, col_id, batch_tokens, hidden_size) / 4;
  } else {
    input_index = i;
  }
  int32_t qinput4 = qdata4[input_index];
  int8_t *qinput = reinterpret_cast<int8_t *>(&qinput4);
  const float4 b4 = __ldg(&bias4[bias_i]);
  const float4 res4 = residual4[i];
  float4 output4;

  output4.x = (dequantize(qinput[0], cmax_val) + b4.x) * scale * m[0] + res4.x;
  output4.y = (dequantize(qinput[1], cmax_val) + b4.y) * scale * m[1] + res4.y;
  output4.z = (dequantize(qinput[2], cmax_val) + b4.z) * scale * m[2] + res4.z;
  output4.w = (dequantize(qinput[3], cmax_val) + b4.w) * scale * m[3] + res4.w;

  out4[i] = output4;
}

__global__ void ls_quant_dropout_res_bias_kernel(
    const int total_count, const float ratio, __half *out, uint8_t *mask,
    const int8_t *qin, const __half *cmax, const __half *bias,
    const __half *residual, const int seed, const int hidden_size,
    bool in_col32) {
  const __half scale = 1.f / (1.f - ratio);

  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 8 >= total_count) return;

  hiprandStatePhilox4_32_10_t state;
  hiprand_init(seed, i, 0, &state);

  const int64_t *qvals8_ptr = reinterpret_cast<const int64_t *>(qin);
  float4 *outs_float4 = reinterpret_cast<float4 *>(out);
  const float4 *residual4 = reinterpret_cast<const float4 *>(residual);
  const float4 *bias4 = reinterpret_cast<const float4 *>(bias);
  uint64_t *mask8 = reinterpret_cast<uint64_t *>(mask);

  uint8_t m[8];
  float4 rand = hiprand_uniform4(&state);
  m[0] = static_cast<uint8_t>(rand.x > ratio);
  m[1] = static_cast<uint8_t>(rand.y > ratio);
  m[2] = static_cast<uint8_t>(rand.z > ratio);
  m[3] = static_cast<uint8_t>(rand.w > ratio);
  rand = hiprand_uniform4(&state);
  m[4] = static_cast<uint8_t>(rand.x > ratio);
  m[5] = static_cast<uint8_t>(rand.y > ratio);
  m[6] = static_cast<uint8_t>(rand.z > ratio);
  m[7] = static_cast<uint8_t>(rand.w > ratio);
  uint64_t *m8 = reinterpret_cast<uint64_t *>(m);
  mask8[i] |= m8[0];

  float cmax_val = __half2float(cmax[0]);
  int bias_i = i % (hidden_size >> 3);
  int input_index;
  if (in_col32) {
    int batch_tokens = total_count / hidden_size;
    int row_id = (i * 8) / hidden_size;
    int col_id = (i * 8) % hidden_size;
    input_index =
        row_major2flat_col32(row_id, col_id, batch_tokens, hidden_size) / 8;
  } else {
    input_index = i;
  }
  int64_t qval8 = qvals8_ptr[input_index];
  int8_t *qval = reinterpret_cast<int8_t *>(&qval8);
  const float4 b4 = __ldg(&bias4[bias_i]);
  const float4 res4 = residual4[i];
  float4 out_float4;

  __half2 *out_half2 = reinterpret_cast<__half2 *>(&out_float4);
  const __half2 *b_half2 = reinterpret_cast<const __half2 *>(&b4);
  const __half2 *res_half2 = reinterpret_cast<const __half2 *>(&res4);

  __half2 scale_mask[4];
  scale_mask[0] =
      __halves2half2(scale * __float2half(m[0]), scale * __float2half(m[1]));
  scale_mask[1] =
      __halves2half2(scale * __float2half(m[2]), scale * __float2half(m[3]));
  scale_mask[2] =
      __halves2half2(scale * __float2half(m[4]), scale * __float2half(m[5]));
  scale_mask[3] =
      __halves2half2(scale * __float2half(m[6]), scale * __float2half(m[7]));

  float2 f_val;
#pragma unroll
  for (int j = 0; j < 4; j++) {
    f_val.x = dequantize(qval[2 * j], cmax_val);
    f_val.y = dequantize(qval[2 * j + 1], cmax_val);
    out_half2[j] = __hfma2(__hadd2(__float22half2_rn(f_val), b_half2[j]),
                           scale_mask[j], res_half2[j]);
  }

  outs_float4[i] = out_float4;
}

template <>
void launch_ls_quant_dropout_res_bias<float>(
    float *out, uint8_t *mask, const int8_t *qvals, const float *cmax,
    const float *bias, const float *residual, int total_count, int dim,
    float ratio, hipStream_t stream, bool in_col32) {
  int grid_dim = total_count >> 12;
  ls_quant_dropout_res_bias_kernel<<<grid_dim + 1, 1024, 0, stream>>>(
      total_count, ratio, out, mask, qvals, cmax, bias, residual,
      std::chrono::duration_cast<std::chrono::microseconds>(
          std::chrono::system_clock::now().time_since_epoch())
          .count(),
      dim, in_col32);
}

template <>
void launch_ls_quant_dropout_res_bias<__half>(
    __half *out, uint8_t *mask, const int8_t *qvals, const __half *cmax,
    const __half *bias, const __half *residual, int total_count, int dim,
    float ratio, hipStream_t stream, bool in_col32) {
  int grid_dim = total_count >> 13;
  ls_quant_dropout_res_bias_kernel<<<grid_dim + 1, 1024, 0, stream>>>(
      total_count, ratio, out, mask, qvals, cmax, bias, residual,
      std::chrono::duration_cast<std::chrono::microseconds>(
          std::chrono::system_clock::now().time_since_epoch())
          .count(),
      dim, in_col32);
}
}  // namespace cuda
}  // namespace lightseq
