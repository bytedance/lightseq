#include "hip/hip_runtime.h"
#include "util.h"

namespace lightseq {
namespace cuda {
struct prg_uniform {
  float a, b;

  __host__ __device__ prg_uniform(float _a = 0.f, float _b = 1.f)
      : a(_a), b(_b){};

  __host__ __device__ float operator()(const unsigned int n) const {
    thrust::default_random_engine rng;
    thrust::uniform_real_distribution<float> dist(a, b);
    rng.discard(n);
    return dist(rng);
  }
};

struct prg_norm {
  float a, b;

  __host__ __device__ prg_norm(float _a = 0.f, float _b = 1.f) : a(_a), b(_b){};

  __host__ __device__ float operator()(const unsigned int n) const {
    thrust::default_random_engine rng;
    thrust::random::normal_distribution<float> dist(a, b);
    rng.discard(n);
    return dist(rng);
  }
};

void generate_distribution(thrust::device_vector<float>& input_output,
                           std::string mode, float a, float b) {
  thrust::counting_iterator<unsigned int> index_sequence_begin(0);
  if (mode == "uniform")
    thrust::transform(index_sequence_begin,
                      index_sequence_begin + input_output.size(),
                      input_output.begin(), prg_uniform(a, b));
  if (mode == "norm")
    thrust::transform(index_sequence_begin,
                      index_sequence_begin + input_output.size(),
                      input_output.begin(), prg_norm(a, b));
}

void read_batch_tokenids_from_file(std::string file_name, int& batch_size,
                                   int& batch_seq_len,
                                   std::vector<int>& input_ids) {
  std::ifstream fin(file_name);
  fin >> batch_size >> batch_seq_len;
  input_ids = std::vector<int>(batch_size * batch_seq_len, 0);
  for (int i = 0; i < batch_size; i++) {
    for (int j = 0; j < batch_seq_len; j++) {
      int idx = i * batch_seq_len + j;
      fin >> input_ids[idx];
    }
  }
}

float dequantize(unsigned char i, float scale, float clip_max) {
  return (float(i) - scale) * clip_max / scale;
}

void dequantize_array(std::vector<unsigned char>& i8, std::vector<float>& f,
                      float clip_max, float quant_range, int start, int num) {
  for (int i = start; i < start + num; ++i) {
    f[i] = dequantize(i8[i], quant_range, clip_max);
  }
}
}  // namespace cuda
}  // namespace lightseq
