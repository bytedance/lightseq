#include "hip/hip_runtime.h"
#include "block_reduce.h"
#include "kernels.h"

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;
const float LN_EPSILON = 1e-8f;
#define TILE_DIM 32

template <typename T>
__forceinline__ __device__ T add_eps(T x) {
  return fabsf(x) > LN_EPSILON ? x : (x < 0 ? -LN_EPSILON : LN_EPSILON);
}

/**
@brief: ker_layer_norm
Standard layer normalization.
It will not only output the layer norm result,
  but also outputs variance.
  may also output means, depends on whether
  the means argument is nullptr

@thread
gridDim.x = batch_size * seq_len
blockDim.x = hidden_size

@param
ln_res: [batch_size* seq_len, hidden_size], ln result.
vars: [batch_size* seq_len], variance per token
means: [batch_size* seq_len], means per token, can be nullput
inp: [batch_size * seq_len, hidden_size], ln input.
scale: [hidden_size], ln scale
bias: [hidden_size], ln bias
*/
template <typename T>
__global__ void ker_layer_norm(T *ln_res, T *vars, T *means, const T *inp,
                               const T *scale, const T *bias, int hidden_size) {
  // step 0. compute local sum
  float l_sum = 0;
  float l_square_sum = 0;
  const float4 *inp_f4 = (const float4 *)inp + blockIdx.x * hidden_size;
  for (uint idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float4 val = inp_f4[idx];
    l_sum += val.x + val.y + val.z + val.w;
    l_square_sum +=
        val.x * val.x + val.y * val.y + val.z * val.z + val.w * val.w;
  }

  // step 1. compute reduce sum
  float mean_dim = float(hidden_size) * 4.f;
  float reduce_val[2] = {l_sum, l_square_sum};
  blockReduce<ReduceType::kSum, 2>(reduce_val);
  __shared__ float s_mean, s_var;
  if (threadIdx.x == 0) {
    s_mean = reduce_val[0] / mean_dim;
    if (means != nullptr) {
      means[blockIdx.x] = s_mean;
    }
    s_var = reduce_val[1] / mean_dim - s_mean * s_mean + LN_EPSILON;
    vars[blockIdx.x] = s_var;
    s_var = rsqrtf(s_var);
  }
  __syncthreads();

  // step 2. layer norm result
  float4 *output_f4 = (float4 *)ln_res + blockIdx.x * hidden_size;
  for (uint idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float4 vscale = __ldg((const float4 *)scale + idx);
    float4 vbias = __ldg((const float4 *)bias + idx);
    float4 val = inp_f4[idx];
    val.x = (val.x - s_mean) * s_var * vscale.x + vbias.x;
    val.y = (val.y - s_mean) * s_var * vscale.y + vbias.y;
    val.z = (val.z - s_mean) * s_var * vscale.z + vbias.z;
    val.w = (val.w - s_mean) * s_var * vscale.w + vbias.w;
    output_f4[idx] = val;
  }
}

template <>
__global__ void ker_layer_norm<__half>(__half *ln_res, __half *vars,
                                       __half *means, const __half *inp,
                                       const __half *scale, const __half *bias,
                                       int hidden_size) {
  // step 0. compute local sum
  float l_sum = 0;
  float l_square_sum = 0;
  const float4 *inp_f4 = (const float4 *)inp + blockIdx.x * hidden_size;
  for (uint idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float4 val_f4 = inp_f4[idx];
    __half2 *val_h2 = (__half2 *)(&val_f4);
#pragma unroll
    for (int i = 0; i < 4; i++) {
      float2 val_f2 = __half22float2(val_h2[i]);
      l_sum += val_f2.x + val_f2.y;
      l_square_sum += val_f2.x * val_f2.x + val_f2.y * val_f2.y;
    }
  }

  // step 1. compute reduce sum
  float mean_dim = float(hidden_size) * 8.f;
  float reduce_val[2] = {l_sum, l_square_sum};
  blockReduce<ReduceType::kSum, 2>(reduce_val);
  __shared__ float s_mean, s_var;
  if (threadIdx.x == 0) {
    s_mean = reduce_val[0] / mean_dim;
    if (means != nullptr) {
      means[blockIdx.x] = s_mean;
    }
    s_var = reduce_val[1] / mean_dim - s_mean * s_mean + LN_EPSILON;
    vars[blockIdx.x] = s_var;
    s_var = rsqrtf(s_var);
  }
  __syncthreads();

  // step 2. layer norm result
  float4 *output_f4 = (float4 *)ln_res + blockIdx.x * hidden_size;
  for (uint idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    // load scale, bias, input
    float4 scale_f4 = __ldg((const float4 *)scale + idx);
    __half2 *scale_h2 = (__half2 *)(&scale_f4);
    float4 bias_f4 = __ldg((const float4 *)bias + idx);
    __half2 *bias_h2 = (__half2 *)(&bias_f4);
    float4 val_f4 = inp_f4[idx];
    __half2 *val_h2 = (__half2 *)(&val_f4);

#pragma unroll
    for (int i = 0; i < 4; i++) {
      float2 scale_f2 = __half22float2(scale_h2[i]);
      float2 bias_f2 = __half22float2(bias_h2[i]);
      float2 val_f2 = __half22float2(val_h2[i]);
      val_f2.x = (val_f2.x - s_mean) * s_var * scale_f2.x + bias_f2.x;
      val_f2.y = (val_f2.y - s_mean) * s_var * scale_f2.y + bias_f2.y;
      val_h2[i] = __float22half2_rn(val_f2);
    }
    output_f4[idx] = val_f4;
  }
}

template <>
void launch_layer_norm<float>(float *ln_res, float *vars, float *means,
                              const float *inp, const float *scale,
                              const float *bias, int batch_size, int hidden_dim,
                              hipStream_t stream) {
  if (hidden_dim % 4 != 0) {
    throw std::runtime_error("violate hidden_dim % 4 = 0");
  }
  hidden_dim >>= 2;
  int nthread = min(((hidden_dim + 31) / 32) * 32, MAX_THREADS);
  dim3 grid_dim(batch_size);
  dim3 block_dim(nthread);

  ker_layer_norm<float><<<grid_dim, block_dim, 0, stream>>>(
      ln_res, vars, means, inp, scale, bias, hidden_dim);
}

template <>
void launch_layer_norm<__half>(__half *ln_res, __half *vars, __half *means,
                               const __half *inp, const __half *scale,
                               const __half *bias, int batch_size,
                               int hidden_dim, hipStream_t stream) {
  if (hidden_dim % 8 != 0) {
    throw std::runtime_error("violate hidden_dim % 8 = 0");
  }
  hidden_dim >>= 3;
  int nthread = min(((hidden_dim + 31) / 32) * 32, MAX_THREADS);
  dim3 grid_dim(batch_size);
  dim3 block_dim(nthread);

  ker_layer_norm<__half><<<grid_dim, block_dim, 0, stream>>>(
      ln_res, vars, means, inp, scale, bias, hidden_dim);
}

/**
@brief: ker_ln_bw_dgamma_dbetta
Layer norm backword kernel, compute the gradient of gamma and betta.
dbetta = sum(dout, dim=0)
dgamma = sum(xhat * dout, dim=0)
xhat = (input - mean) * rsqrt(var) or
  (output - betta) / gamma


@thread
gridDim.x = hidden_size / 32
blockDim.x = 32
blockDim.y = 32

@param
gamma_grad: [hidden_size], gradient of gamma
betta_grad: [hidden_size], gradient of betta
out_grad: [batch_size * seq_len, hidden_size], gradient of betta ln output
inp_or_out: [batch_size * seq_len, hidden_size], ln output if means is nullptr
  ln input if means is not nullptr
gamma: [hidden_size], gamma of ln,
  used to compute xhat, maybe nullptr
betta: [hidden_size], betta of ln,
  used to compute xhat, maybe nullptr
vars: [batch_size * seq_len], variance of ln forward,
  used to compute xhat, maybe nullptr
means: [batch_size * seq_len], mean of ln forward,
  used to compute xhat, maybe nullptr
(gamma && betta) ^ (vars && means) should be true
*/
template <typename T>
__global__ void ker_ln_bw_dgamma_dbetta(T *gamma_grad, T *betta_grad,
                                        const T *out_grad, const T *inp_or_out,
                                        const T *gamma, const T *betta,
                                        const T *vars, const T *means, int rows,
                                        int width) {
  __shared__ float betta_buffer[TILE_DIM][TILE_DIM];
  __shared__ float gamma_buffer[TILE_DIM][TILE_DIM];

  cg::thread_block b = cg::this_thread_block();
  cg::thread_block_tile<TILE_DIM> g = cg::tiled_partition<TILE_DIM>(b);

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int offset = threadIdx.y * width + idx;
  int y_stride = width * TILE_DIM;

  // Loop across inp height
  float dbetta = 0;
  float dgamma = 0;
  float dout, val;
  if (means == nullptr) {
    float vbetta = (float)betta[idx];
    float vgamma = (float)gamma[idx];
    for (int r = threadIdx.y; r < rows; r += TILE_DIM) {
      dout = (float)out_grad[offset];
      // inp_or_out is output
      val = (float)inp_or_out[offset];
      dbetta += dout;
      dgamma += ((val - vbetta) / add_eps(vgamma) * dout);
      offset += y_stride;
    }
  } else {
    for (int r = threadIdx.y; r < rows; r += TILE_DIM) {
      dout = (float)out_grad[offset];
      // inp_or_out is input
      val = (float)inp_or_out[offset];
      dbetta += dout;
      dgamma += ((val - (float)means[r]) * rsqrtf((float)vars[r] + LN_EPSILON) *
                 dout);
      offset += y_stride;
    }
  }

  // Sum the shared buffer.
  betta_buffer[threadIdx.x][threadIdx.y] = dbetta;
  gamma_buffer[threadIdx.x][threadIdx.y] = dgamma;
  __syncthreads();
  float s1 = betta_buffer[threadIdx.y][threadIdx.x];
  float s2 = gamma_buffer[threadIdx.y][threadIdx.x];
  __syncthreads();

  for (int i = 1; i < TILE_DIM; i <<= 1) {
    s1 += g.shfl_down(s1, i);
    s2 += g.shfl_down(s2, i);
  }

  if (threadIdx.x == 0) {
    int pos = blockIdx.x * TILE_DIM + threadIdx.y;
    betta_grad[pos] = s1;
    gamma_grad[pos] = s2;
  }
}

/**
@brief: ker_ln_bw_dinp
Layer norm backword kernel, compute the gradient of input.
dinp = (dxhat - (sum(dxhat) + xhat * sum(dxhat * xhat)) / hidden_dim)
  * rsqrt(var)
xhat = (input - mean) * rsqrt(var) if mean is not nullptr
       (output - betta) / gamma if mean is nullptr
dxhat = dout * gamma


@thread
gridDim.x = batch_size * seq_len
blockDim.x = hidden_size

@param
inp_grad: [batch_size * seq_len, hidden_size], gradient of betta ln output
out_grad: [batch_size * seq_len, hidden_size], gradient of betta ln output
residual_grad: [batch_size * seq_len, hidden_size], gradient of residual input,
  usually appear in pre-layer-norm for transformer layer, maybe nullptr
inp_or_out: [batch_size * seq_len, hidden_size], ln output if means is nullptr
  ln input if means is not nullptr
gamma: [hidden_size], gamma of ln,
  used to compute xhat and dxhat
betta: [hidden_size], betta of ln,
  used to compute xhat, maybe nullptr
vars: [batch_size * seq_len], variance of ln forward,
  used to compute xhat and dinp
means: [batch_size * seq_len], mean of ln forward,
  used to compute xhat, maybe nullptr
*/
template <typename T>
__global__ void ker_ln_bw_dinp(T *inp_grad, const T *out_grad,
                               const T *residual_grad, const T *inp_or_out,
                               const T *gamma, const T *betta, const T *vars,
                               const T *means) {
  float hidden_dim = blockDim.x * 4;
  int offset = blockIdx.x * blockDim.x + threadIdx.x;
  float var_rsqrt = rsqrtf((float)vars[blockIdx.x] + LN_EPSILON);

  // step 0. dxhat = dout * gamma
  float4 dxhat = ((const float4 *)out_grad)[offset];
  float4 vgamma = ((const float4 *)gamma)[threadIdx.x];
  dxhat.x *= vgamma.x;
  dxhat.y *= vgamma.y;
  dxhat.z *= vgamma.z;
  dxhat.w *= vgamma.w;

  /*
  step 1. xhat = (output - betta) / gamma or
  (input - mean) * rsqrtf(var)
  */
  float4 xhat = ((const float4 *)inp_or_out)[offset];
  if (means == nullptr) {
    // inp_or_out is output, xhat = (output - betta) / gamma
    float4 vbetta = ((const float4 *)betta)[threadIdx.x];
    xhat.x = (xhat.x - vbetta.x) / add_eps(vgamma.x);
    xhat.y = (xhat.y - vbetta.y) / add_eps(vgamma.y);
    xhat.z = (xhat.z - vbetta.z) / add_eps(vgamma.z);
    xhat.w = (xhat.w - vbetta.w) / add_eps(vgamma.w);
  } else {
    // inp_or_out is input, xhat = (input - mean) * rsqrtf(var)
    float fmean = (float)means[blockIdx.x];
    xhat.x = (xhat.x - fmean) * var_rsqrt;
    xhat.y = (xhat.y - fmean) * var_rsqrt;
    xhat.z = (xhat.z - fmean) * var_rsqrt;
    xhat.w = (xhat.w - fmean) * var_rsqrt;
  }

  /* step2. block reduce sum for dxhat and dxhat*xhat */
  float sum_dxhat = dxhat.x + dxhat.y + dxhat.z + dxhat.w;
  float sum_dxhat_xhat =
      dxhat.x * xhat.x + dxhat.y * xhat.y + dxhat.z * xhat.z + dxhat.w * xhat.w;
  float reduce_val[2] = {sum_dxhat, sum_dxhat_xhat};
  blockReduce<ReduceType::kSum, 2>(reduce_val);
  __shared__ float s_sum_dxhat, s_sum_dxhat_xhat;
  if (threadIdx.x == 0) {
    s_sum_dxhat = reduce_val[0] / hidden_dim;
    s_sum_dxhat_xhat = reduce_val[1] / hidden_dim;
  }
  __syncthreads();

  /*
  step3. compute input gradient
  (dxhat - (sum(dxhat) + xhat * sum(dxhat * xhat)) / hidden_dim) * rsqrt(var)
  */
  dxhat.x = (dxhat.x - s_sum_dxhat - xhat.x * s_sum_dxhat_xhat) * var_rsqrt;
  dxhat.y = (dxhat.y - s_sum_dxhat - xhat.y * s_sum_dxhat_xhat) * var_rsqrt;
  dxhat.z = (dxhat.z - s_sum_dxhat - xhat.z * s_sum_dxhat_xhat) * var_rsqrt;
  dxhat.w = (dxhat.w - s_sum_dxhat - xhat.w * s_sum_dxhat_xhat) * var_rsqrt;
  if (residual_grad) {
    // Add the residual grad,
    // usually in pre-layer-norm for transformer layer
    float4 dresidual = ((const float4 *)residual_grad)[offset];
    dxhat.x += dresidual.x;
    dxhat.y += dresidual.y;
    dxhat.z += dresidual.z;
    dxhat.w += dresidual.w;
  }
  ((float4 *)inp_grad)[offset] = dxhat;
}

template <>
__global__ void ker_ln_bw_dinp<__half>(__half *inp_grad, const __half *out_grad,
                                       const __half *residual_grad,
                                       const __half *inp_or_out,
                                       const __half *gamma, const __half *betta,
                                       const __half *vars,
                                       const __half *means) {
  float hidden_dim = blockDim.x * 8;
  int offset = blockIdx.x * blockDim.x + threadIdx.x;
  float var_rsqrt = rsqrtf((float)vars[blockIdx.x] + LN_EPSILON);

  // step 0. dxhat = dout * gamma
  float4 vtmp = ((const float4 *)out_grad)[offset];
  __half2 *tmp_h2 = reinterpret_cast<__half2 *>(&vtmp);
  float4 vgamma = ((const float4 *)gamma)[threadIdx.x];
  __half2 *gamma_h2 = reinterpret_cast<__half2 *>(&vgamma);
  float2 dxhat[4];
  float sum_dxhat = 0;
#pragma unroll
  for (int i = 0; i < 4; i++) {
    float2 vdout = __half22float2(tmp_h2[i]);
    float2 vgamma = __half22float2(gamma_h2[i]);
    dxhat[i].x = vdout.x * vgamma.x;
    dxhat[i].y = vdout.y * vgamma.y;
    sum_dxhat += dxhat[i].x + dxhat[i].y;
  }

  /*
  step 1. xhat = (output - betta) / gamma or
  (input - mean) * rsqrtf(var)
  */
  vtmp = ((const float4 *)inp_or_out)[offset];
  float2 xhat[4];
  float sum_dxhat_xhat = 0;
  if (means == nullptr) {
    // inp_or_out is output, xhat = (output - betta) / gamma
    float4 vbetta = ((const float4 *)betta)[threadIdx.x];
    __half2 *betta_h2 = reinterpret_cast<__half2 *>(&vbetta);
#pragma unroll
    for (int i = 0; i < 4; i++) {
      float2 vout = __half22float2(tmp_h2[i]);
      float2 vgamma = __half22float2(gamma_h2[i]);
      float2 vbetta = __half22float2(betta_h2[i]);
      xhat[i].x = (vout.x - vbetta.x) / add_eps(vgamma.x);
      xhat[i].y = (vout.y - vbetta.y) / add_eps(vgamma.y);
      sum_dxhat_xhat += xhat[i].x * dxhat[i].x + xhat[i].y * dxhat[i].y;
    }
  } else {
    // inp_or_out is input, xhat = (input - mean) * rsqrtf(var)
    float fmean = (float)means[blockIdx.x];
#pragma unroll
    for (int i = 0; i < 4; i++) {
      float2 vinp = __half22float2(tmp_h2[i]);
      xhat[i].x = (vinp.x - fmean) * var_rsqrt;
      xhat[i].y = (vinp.y - fmean) * var_rsqrt;
      sum_dxhat_xhat += xhat[i].x * dxhat[i].x + xhat[i].y * dxhat[i].y;
    }
  }

  /* step2. block reduce sum for dxhat and dxhat*xhat */
  float reduce_val[2] = {sum_dxhat, sum_dxhat_xhat};
  blockReduce<ReduceType::kSum, 2>(reduce_val);
  __shared__ float s_sum_dxhat, s_sum_dxhat_xhat;
  if (threadIdx.x == 0) {
    s_sum_dxhat = reduce_val[0] / hidden_dim;
    s_sum_dxhat_xhat = reduce_val[1] / hidden_dim;
  }
  __syncthreads();

  /*
  step3. compute input gradient
  (dxhat - (sum(dxhat) + xhat * sum(dxhat * xhat)) / hidden_dim) * rsqrt(var)
  */
  if (residual_grad) {
    // Add the residual grad,
    // usually in pre-layer-norm for transformer layer
    float4 dresidual = ((const float4 *)residual_grad)[offset];
    __half *hdres = reinterpret_cast<__half *>(&dresidual);
#pragma unroll
    for (int i = 0; i < 4; i++) {
      tmp_h2[i].x = __float2half(
          (dxhat[i].x - s_sum_dxhat - xhat[i].x * s_sum_dxhat_xhat) *
              var_rsqrt +
          __half2float(hdres[2 * i]));
      tmp_h2[i].y = __float2half(
          (dxhat[i].y - s_sum_dxhat - xhat[i].y * s_sum_dxhat_xhat) *
              var_rsqrt +
          __half2float(hdres[2 * i + 1]));
    }
  } else {
#pragma unroll
    for (int i = 0; i < 4; i++) {
      tmp_h2[i].x = __float2half(
          (dxhat[i].x - s_sum_dxhat - xhat[i].x * s_sum_dxhat_xhat) *
          var_rsqrt);
      tmp_h2[i].y = __float2half(
          (dxhat[i].y - s_sum_dxhat - xhat[i].y * s_sum_dxhat_xhat) *
          var_rsqrt);
    }
  }
  ((float4 *)inp_grad)[offset] = vtmp;
}

/**
Layer norm backword,
  compute the gradient of gamma, betta and input.
dbetta = sum(dout, dim=0)
xhat = (input - mean) * rsqrt(var) if mean is not nullptr
  (output - betta) / gamma if mean is nullptr
dgamma = sum(xhat * dout, dim=0)
dxhat = dout * gamma
dinp = (dxhat - (sum(dxhat, 1) + xhat * sum(dxhat * xhat, 1)) / hidden_dim)
  * rsqrt(var)

residual_grad, means, betta can be nullptr.
residual_grad will be added to dinp if it is not nullptr
  which is useful in transformer layer when pre-ln
means and betta are only used to compute xhat,
  (means == nullptr) ^ (betta == nullptr) should be true
*/
template <>
void launch_ln_bw<float>(float *gamma_grad, float *betta_grad, float *inp_grad,
                         const float *out_grad, const float *residual_grad,
                         const float *inp_or_out, const float *gamma,
                         const float *betta, const float *vars,
                         const float *means, int batch, int hidden_dim,
                         hipStream_t stream[2]) {
  // compute grad of gamma and betta
  dim3 grid_dim(hidden_dim / TILE_DIM);
  dim3 block_dim(TILE_DIM, TILE_DIM);
  ker_ln_bw_dgamma_dbetta<float><<<grid_dim, block_dim, 0, stream[0]>>>(
      gamma_grad, betta_grad, out_grad, inp_or_out, gamma, betta, vars, means,
      batch, hidden_dim);

  // compute grad of input
  hidden_dim >>= 2;
  ker_ln_bw_dinp<<<batch, hidden_dim, 0, stream[1]>>>(
      inp_grad, out_grad, residual_grad, inp_or_out, gamma, betta, vars, means);
}

template <>
void launch_ln_bw<__half>(__half *gamma_grad, __half *betta_grad,
                          __half *inp_grad, const __half *out_grad,
                          const __half *residual_grad, const __half *inp_or_out,
                          const __half *gamma, const __half *betta,
                          const __half *vars, const __half *means, int batch,
                          int hidden_dim, hipStream_t stream[2]) {
  // compute grad of gamma and betta
  dim3 grid_dim(hidden_dim / TILE_DIM);
  dim3 block_dim(TILE_DIM, TILE_DIM);
  ker_ln_bw_dgamma_dbetta<__half><<<grid_dim, block_dim, 0, stream[0]>>>(
      gamma_grad, betta_grad, out_grad, inp_or_out, gamma, betta, vars, means,
      batch, hidden_dim);

  // compute grad of input
  hidden_dim >>= 3;
  ker_ln_bw_dinp<<<batch, hidden_dim, 0, stream[1]>>>(
      inp_grad, out_grad, residual_grad, inp_or_out, gamma, betta, vars, means);
}
