#include "hip/hip_runtime.h"
#include <random>

#include "common.h"
#include "gptKernels.h"
#include "transformerKernels.h"
/**
@file
Implemented the cuda kernel function and its launcher
that required by GPT model.
Currently, fp16 and fp32 versions are provided
*/
namespace lightseq {
namespace cuda {

/**
@brief: ker_gpt_embedding
for encoder, look up token embedding, add position embedding

@thread
gridDim.x = batch_size
gridDim.y = token_seq_len
blockDim.x = hidden_size

@param
token_emb: [vocab_size, hidden_size]
pos_emb: [max_step, hidden_size]
token_id: input token id, [batch_size, token_seq_len]
output: result, [batch_size, token_seq_len, hidden_size]
real_seq_len: record seq len exclude padding, [batch_size]
padding_id, the padding_id, default 0
pos_offset: get real pos when decoding which gridDim.y=1
*/
template <typename T>
__global__ void ker_gpt_embedding(const T* token_emb, const T* pos_emb,
                                  const int* token_id, T* output,
                                  int* real_seq_len, int padding_id,
                                  int pos_offset) {
  int target_pos = blockIdx.x * gridDim.y + blockIdx.y;
  int tid = token_id[target_pos];
  if (tid == padding_id) {
    // for padding id
    output[target_pos * blockDim.x + threadIdx.x] = 0.f;
    return;
  }
  if (threadIdx.x == 0) {
    atomicAdd(real_seq_len + blockIdx.x, 1);
  }
  output[target_pos * blockDim.x + threadIdx.x] =
      token_emb[tid * blockDim.x + threadIdx.x] +
      pos_emb[(blockIdx.y + pos_offset) * blockDim.x + threadIdx.x];
}

/* fp16 version */
template <>
__global__ void ker_gpt_embedding<__half>(const __half* token_emb,
                                          const __half* pos_emb,
                                          const int* token_id, __half* output,
                                          int* real_seq_len, int padding_id,
                                          int pos_offset) {
  int target_pos = blockIdx.x * gridDim.y + blockIdx.y;
  int tid = token_id[target_pos];
  half2* output_h = (half2*)output;

  if (tid == padding_id) {
    // for padding id
    output_h[target_pos * blockDim.x + threadIdx.x] = __float2half2_rn(0.f);
    return;
  }
  if (threadIdx.x == 0) {
    atomicAdd(real_seq_len + blockIdx.x, 1);
  }

  float2 te =
      __half22float2(((const half2*)token_emb)[tid * blockDim.x + threadIdx.x]);
  float2 pe = __half22float2(
      ((const half2*)
           pos_emb)[(blockIdx.y + pos_offset) * blockDim.x + threadIdx.x]);
  te.x += pe.x;
  te.y += pe.y;
  output_h[target_pos * blockDim.x + threadIdx.x] = __float22half2_rn(te);
}

template <typename T>
void ker_gpt_embedding_launcher(int batch_size, int batch_seq_len,
                                int hidden_size, hipStream_t stream,
                                const T* token_emb, const T* pos_emb,
                                const int* token_id, T* output,
                                int* real_seq_len, int padding_id,
                                int pos_offset) {
  ker_gpt_embedding<T>
      <<<dim3(batch_size, batch_seq_len), hidden_size, 0, stream>>>(
          token_emb, pos_emb, token_id, output, real_seq_len, padding_id,
          pos_offset);
}

template <>
void ker_gpt_embedding_launcher<__half>(
    int batch_size, int batch_seq_len, int hidden_size, hipStream_t stream,
    const __half* token_emb, const __half* pos_emb, const int* token_id,
    __half* output, int* real_seq_len, int padding_id, int pos_offset) {
  ker_gpt_embedding<__half>
      <<<dim3(batch_size, batch_seq_len), hidden_size / 2, 0, stream>>>(
          token_emb, pos_emb, token_id, output, real_seq_len, padding_id,
          pos_offset);
}

template void ker_gpt_embedding_launcher<float>(
    int batch_size, int batch_seq_len, int hidden_size, hipStream_t stream,
    const float* token_emb, const float* pos_emb, const int* token_id,
    float* output, int* real_seq_len, int padding_id, int pos_offset);

template void ker_gpt_embedding_launcher<__half>(
    int batch_size, int batch_seq_len, int hidden_size, hipStream_t stream,
    const __half* token_emb, const __half* pos_emb, const int* token_id,
    __half* output, int* real_seq_len, int padding_id, int pos_offset);

/**
@brief: ker_correlation_softmax_gpt
query-key correlation softmax for encoder self attention

@thread
gridDim.x = batch_size
gridDim.y = head_num * batch_seq_len
blockDim.x = batch_seq_len

@param
correlation: [batch_size, head_num, batch_seq_len, batch_seq_len]
real_seq_len: [batch_size]
*/
template <typename T>
__global__ void ker_correlation_softmax_gpt(T* correlation,
                                            const int* real_seq_len,
                                            const int batch_seq_len) {
  int query_token_pos = blockIdx.y % batch_seq_len;
  if (query_token_pos >= real_seq_len[blockIdx.x]) {
    return;
  }

  int mask = 0;  // can see the token when mask=0
  if (threadIdx.x > query_token_pos || threadIdx.x >= batch_seq_len) {
    mask = 1;  // Can only see the token on the left side of it
  }

  int idx = (blockIdx.x * gridDim.y + blockIdx.y) * batch_seq_len + threadIdx.x;
  float val = threadIdx.x < batch_seq_len ? (float)correlation[idx]
                                          : CUDA_FLOAT_INF_NEG;
  float max_val = blockReduceMax<float>(mask ? CUDA_FLOAT_INF_NEG : val);
  __shared__ float smax;
  if (threadIdx.x == 0) smax = max_val;
  __syncthreads();

  val = mask ? 0.f : expf(val - smax);
  float rsum = blockReduceSum<float>(val);
  __shared__ float ssum;
  if (threadIdx.x == 0) ssum = rsum;
  __syncthreads();

  if (threadIdx.x < batch_seq_len) correlation[idx] = (T)(val / ssum);
}

template <typename T>
void ker_correlation_softmax_gpt_launcher(int batch_size, int batch_seq_len,
                                          int head_num, hipStream_t stream,
                                          T* correlation,
                                          const int* real_seq_len) {
  int block_dim = batch_seq_len;
  if (batch_seq_len < 1024) {
    block_dim = (batch_seq_len + 31) >> 5;
    block_dim *= 32;
  }

  ker_correlation_softmax_gpt<T>
      <<<dim3(batch_size, head_num * batch_seq_len), block_dim, 0, stream>>>(
          correlation, real_seq_len, batch_seq_len);
}

template void ker_correlation_softmax_gpt_launcher<float>(
    int batch_size, int batch_seq_len, int head_num, hipStream_t stream,
    float* correlation, const int* real_seq_len);

template void ker_correlation_softmax_gpt_launcher<__half>(
    int batch_size, int batch_seq_len, int head_num, hipStream_t stream,
    __half* correlation, const int* real_seq_len);

/**
@brief: ker_attention_mask_weights
query-key correlation softmax for encoder self attention

@thread
gridDim.x = batch_size
gridDim.y = head_num * dst_seq_len
blockDim.x = src_seq_len

@param
correlation: [batch_size, head_num, dst_seq_len, src_seq_len]
real_seq_len: [batch_size]
*/
template <typename T>
__global__ void ker_attention_mask_weights(T* correlation,
                                           const int* real_seq_len,
                                           int dst_seq_len, int src_seq_len) {
  int query_token_pos = blockIdx.y % dst_seq_len + src_seq_len - dst_seq_len;
  if (query_token_pos >= real_seq_len[blockIdx.x]) {
    return;
  }
  int mask = 0;  // can see the token when mask=0
  if (threadIdx.x > query_token_pos) {
    mask = 1;  // Can only see the token on the left side of it
  }

  int idx = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  float val = (float)correlation[idx];
  float max_val = blockReduceMax<float>(mask ? CUDA_FLOAT_INF_NEG : val);
  __shared__ float smax;
  if (threadIdx.x == 0) smax = max_val;
  __syncthreads();

  val = mask ? 0.f : expf(fmaxf(logit_thresh_min, val - smax));
  float rsum = blockReduceSum<float>(val);
  __shared__ float ssum;
  if (threadIdx.x == 0) ssum = rsum;
  __syncthreads();

  correlation[idx] = (T)(val / (ssum + epsilon));
}

template <typename T>
void ker_attention_mask_weights_launcher(int batch_size, int dst_seq_len,
                                         int src_seq_len, int head_num,
                                         hipStream_t stream, T* correlation,
                                         const int* real_seq_len) {
  ker_attention_mask_weights<T>
      <<<dim3(batch_size, head_num * dst_seq_len), src_seq_len, 0, stream>>>(
          correlation, real_seq_len, dst_seq_len, src_seq_len);
}

template void ker_attention_mask_weights_launcher<float>(
    int batch_size, int dst_seq_len, int src_seq_len, int head_num,
    hipStream_t stream, float* correlation, const int* real_seq_len);

template void ker_attention_mask_weights_launcher<__half>(
    int batch_size, int dst_seq_len, int src_seq_len, int head_num,
    hipStream_t stream, __half* correlation, const int* real_seq_len);

/**
@brief: ker_arrange_qkv_with_cache
split and reshape ori_qkv matrix into new_q, new_k, new_v during encoder
self-attention
ori_qkv is the result of gemm

@thread
gridDim.x = batch_size * batch_seq_len
gridDim.y = 3
blockDim.x = hidden_size

@param
ori_qkv: [batch_size, 1, 3, hidden_size]
qkv_bias: [3, hidden_size]
new_q: [batch_size, head_num, 1, dim_per_head]
max_batch_dim: max_batch_size * max_seq_len * hidden_size
batch_seq_len: the sequence length of the current batch
dim_per_head: dim of one head in multi-head attention
head_num: head number in multi-head attention
*/
template <typename T>
__global__ void ker_arrange_qkv_with_cache(const T* ori_qkv, const T* qkv_bias,
                                           T* new_q, T* new_k, T* k_cache,
                                           T* new_v, T* v_cache,
                                           int max_batch_dim, int batch_seq_len,
                                           int dim_per_head, int head_num) {
  int batch_id = blockIdx.x / batch_seq_len;
  int token_id = blockIdx.x % batch_seq_len;
  int head_id = threadIdx.x / dim_per_head;
  int dim_id = threadIdx.x % dim_per_head;
  int target_id = targetid_4dim(batch_id, head_id, token_id, dim_id, head_num,
                                batch_seq_len, dim_per_head);
  T new_val;

  if (token_id < batch_seq_len - 1) {
    int old_target_id =
        targetid_4dim(batch_id, head_id, token_id, dim_id, head_num,
                      batch_seq_len - 1, dim_per_head);
    if (blockIdx.y == 0) return;
    if (blockIdx.y == 1) new_val = k_cache[old_target_id];
    if (blockIdx.y == 2) new_val = v_cache[old_target_id];
  } else {
    new_val = ori_qkv[(batch_id * gridDim.y + blockIdx.y) * blockDim.x +
                      threadIdx.x] +
              __ldg(&qkv_bias[blockIdx.y * blockDim.x + threadIdx.x]);
    if (blockIdx.y == 0) {
      target_id = targetid_4dim(batch_id, head_id, 0, dim_id, head_num, 1,
                                dim_per_head);
    }
  }

  if (blockIdx.y == 0) new_q[target_id] = new_val;
  if (blockIdx.y == 1) new_k[target_id] = new_val;
  if (blockIdx.y == 2) new_v[target_id] = new_val;
}

template <>
__global__ void ker_arrange_qkv_with_cache<__half>(
    const __half* ori_qkv, const __half* qkv_bias, __half* new_q, __half* new_k,
    __half* k_cache, __half* new_v, __half* v_cache, int max_batch_dim,
    int batch_seq_len, int dim_per_head, int head_num) {
  int batch_id = blockIdx.x / batch_seq_len;
  int token_id = blockIdx.x % batch_seq_len;
  int head_id = threadIdx.x / dim_per_head;
  int dim_id = threadIdx.x % dim_per_head;
  int target_id = targetid_4dim(batch_id, head_id, token_id, dim_id, head_num,
                                batch_seq_len, dim_per_head);
  half2 new_val;
  const half2* p_ori_qkv = (const half2*)ori_qkv;
  const half2* p_bias = (const half2*)qkv_bias;
  const half2* p_k_cache = (const half2*)k_cache;
  const half2* p_v_cache = (const half2*)v_cache;
  half2* p_new_q = (half2*)new_q;
  half2* p_new_k = (half2*)new_k;
  half2* p_new_v = (half2*)new_v;

  if (token_id < batch_seq_len - 1) {
    int old_target_id =
        targetid_4dim(batch_id, head_id, token_id, dim_id, head_num,
                      batch_seq_len - 1, dim_per_head);
    if (blockIdx.y == 0) return;
    if (blockIdx.y == 1) new_val = p_k_cache[old_target_id];
    if (blockIdx.y == 2) new_val = p_v_cache[old_target_id];
  } else {
    new_val =
        __hadd2(p_ori_qkv[(batch_id * gridDim.y + blockIdx.y) * blockDim.x +
                          threadIdx.x],
                __ldg(&p_bias[blockIdx.y * blockDim.x + threadIdx.x]));
    if (blockIdx.y == 0) {
      target_id = targetid_4dim(batch_id, head_id, 0, dim_id, head_num, 1,
                                dim_per_head);
    }
  }

  if (blockIdx.y == 0) p_new_q[target_id] = new_val;
  if (blockIdx.y == 1) p_new_k[target_id] = new_val;
  if (blockIdx.y == 2) p_new_v[target_id] = new_val;
}

template <typename T>
void ker_arrange_qkv_with_cache_launcher(int batch_token_num, int hidden_size,
                                         hipStream_t stream, const T* ori_qkv,
                                         const T* qkv_bias, T* new_q, T* new_k,
                                         T* k_cache, T* new_v, T* v_cache,
                                         int max_batch_dim, int batch_seq_len,
                                         int dim_per_head, int head_num) {
  ker_arrange_qkv_with_cache<T>
      <<<dim3(batch_token_num, 3), hidden_size, 0, stream>>>(
          ori_qkv, qkv_bias, new_q, new_k, k_cache, new_v, v_cache,
          max_batch_dim, batch_seq_len, dim_per_head, head_num);
}

template <>
void ker_arrange_qkv_with_cache_launcher<__half>(
    int batch_token_num, int hidden_size, hipStream_t stream,
    const __half* ori_qkv, const __half* qkv_bias, __half* new_q, __half* new_k,
    __half* k_cache, __half* new_v, __half* v_cache, int max_batch_dim,
    int batch_seq_len, int dim_per_head, int head_num) {
  ker_arrange_qkv_with_cache<__half>
      <<<dim3(batch_token_num, 3), hidden_size / 2, 0, stream>>>(
          ori_qkv, qkv_bias, new_q, new_k, k_cache, new_v, v_cache,
          max_batch_dim / 2, batch_seq_len, dim_per_head / 2, head_num);
}

template void ker_arrange_qkv_with_cache_launcher<float>(
    int batch_token_num, int hidden_size, hipStream_t stream,
    const float* ori_qkv, const float* qkv_bias, float* new_q, float* new_k,
    float* k_cache, float* new_v, float* v_cache, int max_batch_dim,
    int batch_seq_len, int dim_per_head, int head_num);

template void ker_arrange_qkv_with_cache_launcher<__half>(
    int batch_token_num, int hidden_size, hipStream_t stream,
    const __half* ori_qkv, const __half* qkv_bias, __half* new_q, __half* new_k,
    __half* k_cache, __half* new_v, __half* v_cache, int max_batch_dim,
    int batch_seq_len, int dim_per_head, int head_num);

/**
@brief: ker_ppl
compute ppl from logit
ppl = - (1 / n) * sum(log(i|i-1...))
one thread block compute log probability for the given token

@thread
gridDim.x = batch_size
gridDim.y = batch_seq_len
blockDim.x = max_thread_per_block

@param
logits: [batch_size, batch_seq_len, vocab_size]
input_ids: [batch_size, batch_seq_len]
real_seq_len: [batch_size]
ppl: [batch_size]
*/
template <typename T>
__global__ void ker_ppl(const T* logits, const int* input_ids,
                        const int* real_seq_len, float* ppl, int vocab_size) {
  int seq_len = real_seq_len[blockIdx.x];  // remove "eos"
  if (blockIdx.y >= seq_len - 1) {
    // will not contribute to ppl
    return;
  }

  int token_idx_in_batch = blockIdx.x * gridDim.y + blockIdx.y;
  int left_logit_idx = token_idx_in_batch * vocab_size + threadIdx.x;
  int right_logit_idx = (token_idx_in_batch + 1) * vocab_size;
  /*
  step 1. find max logit over the whole vocab
  */
  float max_logit = CUDA_FLOAT_INF_NEG;
  for (int idx = left_logit_idx; idx < right_logit_idx; idx += blockDim.x) {
    max_logit = fmaxf(max_logit, (float)logits[idx]);
  }
  max_logit = blockReduceMax(max_logit);
  __shared__ float s_max_logit;
  if (threadIdx.x == 0) {
    s_max_logit = max_logit;
  }
  __syncthreads();

  /*
  step 2. compute the log probability for the given token,
  add it to the sequence's ppl
  */
  float sum_exp_logit = 0.f;
  for (int idx = left_logit_idx; idx < right_logit_idx; idx += blockDim.x) {
    float lgt = fmaxf((float)logits[idx] - s_max_logit, logit_thresh_min);
    sum_exp_logit += expf(lgt);
  }
  sum_exp_logit = blockReduceSum(sum_exp_logit);

  if (threadIdx.x == 0) {
    int token_id = input_ids[token_idx_in_batch + 1];
    float log_prob =
        ((float)logits[token_idx_in_batch * vocab_size + token_id] -
         s_max_logit - logf(sum_exp_logit)) /
        (float)(seq_len - 1);
    atomicAdd(ppl + blockIdx.x, -log_prob);
  }
}

template <typename T>
void ker_ppl_launcher(int batch_size, int batch_seq_len,
                      int max_thread_per_block, hipStream_t stream,
                      const T* logits, const int* input_ids,
                      const int* real_seq_len, float* ppl, int vocab_size) {
  ker_ppl<T>
      <<<dim3(batch_size, batch_seq_len), max_thread_per_block, 0, stream>>>(
          logits, input_ids, real_seq_len, ppl, vocab_size);
}

template void ker_ppl_launcher<float>(int batch_size, int batch_seq_len,
                                      int max_thread_per_block,
                                      hipStream_t stream, const float* logits,
                                      const int* input_ids,
                                      const int* real_seq_len, float* ppl,
                                      int vocab_size);

template void ker_ppl_launcher<__half>(
    int batch_size, int batch_seq_len, int max_thread_per_block,
    hipStream_t stream, const __half* logits, const int* input_ids,
    const int* real_seq_len, float* ppl, int vocab_size);

/**
@brief: ker_topk_sample

@thread
gridDim.x = batch_size
blockDim.x = max_thread_per_block

@param
logits: [batch_size, logits_seq_len, vocab_size]
old_input_ids: [batch_size, batch_seq_len]
new_input_ids: [batch_size, batch_seq_len+1]
real_seq_len: [batch_size]
unfinished: [1]
curandstate: [batch_size]
*/
template <typename T, int k>
__global__ void ker_topk_sample(const T* logits, int* old_input_ids,
                                int* new_input_ids, const int* real_seq_len,
                                const int vocab_size, const int batch_seq_len,
                                int logits_seq_len, int* unfinished,
                                hiprandState* curandstate, int eos_id) {
  int last_token_idx_in_batch = blockIdx.x * batch_seq_len + batch_seq_len - 1;

  /* add EOS to end if last token is EOS */
  if (old_input_ids[last_token_idx_in_batch] == eos_id) {
    int left_token_idx = blockIdx.x * batch_seq_len + threadIdx.x;
    int right_token_idx = (blockIdx.x + 1) * batch_seq_len;
    for (int idx = left_token_idx; idx < right_token_idx; idx += blockDim.x) {
      int new_idx = idx + blockIdx.x;
      new_input_ids[new_idx] = old_input_ids[idx];
    }
    if (threadIdx.x == 0) {
      // blockIdx.x * (batch_seq_len+1) + batch_seq_len
      new_input_ids[(blockIdx.x + 1) * (batch_seq_len + 1) - 1] = eos_id;
      old_input_ids[gridDim.x * batch_seq_len + blockIdx.x] = eos_id;
    }
    return;
  }
  int logits_token_idx_in_batch =
      blockIdx.x * logits_seq_len + logits_seq_len - 1;
  int left_logit_idx = logits_token_idx_in_batch * vocab_size + threadIdx.x;
  int right_logit_idx = (logits_token_idx_in_batch + 1) * vocab_size;

  /*
  step1. find max logit and rough Kth logit over the whole vocab
  */
  __shared__ float s_max_logit, s_topk_logit;
  float rough_top_kth_logit = CUDA_FLOAT_INF_NEG;
  for (int idx = left_logit_idx; idx < right_logit_idx; idx += blockDim.x) {
    rough_top_kth_logit = fmaxf(rough_top_kth_logit, (float)logits[idx]);
  }
  float max_logit = blockReduceMax(rough_top_kth_logit);
  rough_top_kth_logit = blockRoughTopK<float, k>(rough_top_kth_logit);
  if (threadIdx.x == 0) {
    s_topk_logit = rough_top_kth_logit;
    s_max_logit = max_logit;
  }
  __syncthreads();

  __shared__ int s_tid;

  if (k != 1) {
    /* step2 hold one logit per thread which larger than Kth logit and sample
     * from them */
    float topk_exp_sum, topk_exp = CUDA_FLOAT_INF_NEG;
    int topk_tid = vocab_size;
    int test_num = 0;
    __shared__ float s_topk_exp_sum;
    for (int idx = left_logit_idx; idx < right_logit_idx; idx += blockDim.x) {
      float logit = (float)logits[idx];
      float logit_exp = expf(fmaxf(logit - s_max_logit, logit_thresh_min));
      if (logit >= s_topk_logit) test_num++;
      if (logit >= s_topk_logit && logit_exp > topk_exp) {
        topk_exp = logit_exp;
        topk_tid = idx - left_logit_idx + threadIdx.x;
      }
    }

    test_num = blockReduceSum(test_num);

    if (topk_tid == vocab_size) topk_exp = 0;
    topk_exp_sum = blockReduceSum(topk_exp);
    if (threadIdx.x == 0) {
      s_topk_exp_sum = topk_exp_sum;
    }
    __syncthreads();

    /* calculate cumulative probability */
    float topk_prob = topk_exp / s_topk_exp_sum;
    float prefix_sum_prob;
    typedef hipcub::BlockScan<float, 1024> BlockScan;
    __shared__ typename BlockScan::TempStorage temp_storage;
    BlockScan(temp_storage).InclusiveSum(topk_prob, prefix_sum_prob);

    __shared__ float random_x;
    if (threadIdx.x == 0) {
      random_x = hiprand_uniform(curandstate + blockIdx.x);
    }
    __syncthreads();

    if (threadIdx.x == 0) {
      s_tid = vocab_size;
    }
    __syncthreads();

    int threadID = threadIdx.x;
    __shared__ int s_threadID;
    __shared__ float s_max_prob;
    if (random_x > prefix_sum_prob) threadID = blockDim.x;
    threadID = blockReduceMin(threadID);
    float max_prob = blockReduceMax(topk_prob);
    if (threadIdx.x == 0) {
      s_threadID = threadID;
      s_max_prob = max_prob;
    }
    __syncthreads();
    if (threadIdx.x == s_threadID) {
      s_tid = topk_tid;
    }
    __syncthreads();

    if (s_tid == vocab_size && topk_prob == s_max_prob) {
      s_tid = topk_tid;
    }
    __syncthreads();
  } else {
    s_tid = vocab_size;
    for (int idx = left_logit_idx; idx < right_logit_idx; idx += blockDim.x) {
      float logit = (float)logits[idx];
      if (logit == s_max_logit) {
        s_tid = idx - left_logit_idx + threadIdx.x;
      }
    }
    __syncthreads();
  }

  /* if new sampled tid is not EOS, set unfinish TRUE */
  if (threadIdx.x == 0) {
    if (s_tid != eos_id) unfinished[0] = 1;
  }

  /* step3 copy old_input_ids to new_input_ids and add new sampled ids */
  int left_token_idx = blockIdx.x * batch_seq_len + threadIdx.x;
  int right_token_idx = (blockIdx.x + 1) * batch_seq_len;
  for (int idx = left_token_idx; idx < right_token_idx; idx += blockDim.x) {
    int new_idx = idx + blockIdx.x;
    new_input_ids[new_idx] = old_input_ids[idx];
  }
  if (threadIdx.x == 0) {
    new_input_ids[(blockIdx.x + 1) * (batch_seq_len + 1) - 1] = s_tid;
    //  save the newly sampled ids to old_input_ids for next step inputs
    old_input_ids[gridDim.x * batch_seq_len + blockIdx.x] = s_tid;
  }
}

template <typename T>
void ker_topk_sample_launcher(int batch_size, int batch_seq_len,
                              int logits_seq_len, int max_thread_per_block,
                              hipStream_t stream, const T* logits,
                              int* old_input_ids, int* new_input_ids,
                              const int* real_seq_len, const int vocab_size,
                              const int k, int* unfinished,
                              hiprandState* curandstate, int eos_id) {
  if (k == 1)
    ker_topk_sample<T, 1><<<batch_size, max_thread_per_block, 0, stream>>>(
        logits, old_input_ids, new_input_ids, real_seq_len, vocab_size,
        batch_seq_len, logits_seq_len, unfinished, curandstate, eos_id);
  else if (k == 2)
    ker_topk_sample<T, 2><<<batch_size, max_thread_per_block, 0, stream>>>(
        logits, old_input_ids, new_input_ids, real_seq_len, vocab_size,
        batch_seq_len, logits_seq_len, unfinished, curandstate, eos_id);
  else if (k == 4)
    ker_topk_sample<T, 4><<<batch_size, max_thread_per_block, 0, stream>>>(
        logits, old_input_ids, new_input_ids, real_seq_len, vocab_size,
        batch_seq_len, logits_seq_len, unfinished, curandstate, eos_id);
  else if (k == 8)
    ker_topk_sample<T, 8><<<batch_size, max_thread_per_block, 0, stream>>>(
        logits, old_input_ids, new_input_ids, real_seq_len, vocab_size,
        batch_seq_len, logits_seq_len, unfinished, curandstate, eos_id);
  else if (k == 16)
    ker_topk_sample<T, 16><<<batch_size, max_thread_per_block, 0, stream>>>(
        logits, old_input_ids, new_input_ids, real_seq_len, vocab_size,
        batch_seq_len, logits_seq_len, unfinished, curandstate, eos_id);
  else if (k == 32)
    ker_topk_sample<T, 32><<<batch_size, max_thread_per_block, 0, stream>>>(
        logits, old_input_ids, new_input_ids, real_seq_len, vocab_size,
        batch_seq_len, logits_seq_len, unfinished, curandstate, eos_id);
  else {
    throw std::invalid_argument("topk argument should be in [1,2,4,8,16,32]");
  }
}

template void ker_topk_sample_launcher<float>(
    int batch_size, int batch_seq_len, int logits_seq_len,
    int max_thread_per_block, hipStream_t stream, const float* logits,
    int* old_input_ids, int* new_input_idx, const int* real_seq_len,
    const int vocab_size, const int k, int* unfinished,
    hiprandState* curandstate, int eos_id);

template void ker_topk_sample_launcher<__half>(
    int batch_size, int batch_seq_len, int logits_seq_len,
    int max_thread_per_block, hipStream_t stream, const __half* logits,
    int* old_input_ids, int* new_input_idx, const int* real_seq_len,
    const int vocab_size, const int k, int* unfinished,
    hiprandState* curandstate, int eos_id);

/**
@brief: ker_topp_sample

@thread
gridDim.x = batch_size
blockDim.x = max_thread_per_block

@param
logits: [batch_size, logits_seq_len, vocab_size]
old_input_ids: [batch_size, batch_seq_len]
new_input_ids: [batch_size, batch_seq_len+1]
real_seq_len: [batch_size]
unfinished: [1]
curandstate: [batch_size]
*/
template <typename T>
__global__ void ker_topp_sample(const T* logits, int* old_input_ids,
                                int* new_input_ids, const int* real_seq_len,
                                const int vocab_size, const int batch_seq_len,
                                int logits_seq_len, int* unfinished, float p,
                                hiprandState* curandstate, int eos_id) {
  int token_idx_in_batch = blockIdx.x * batch_seq_len + batch_seq_len - 1;

  /* add EOS to end if last token is EOS */
  if (old_input_ids[token_idx_in_batch] == eos_id) {
    int left_token_idx = blockIdx.x * batch_seq_len + threadIdx.x;
    int right_token_idx = (blockIdx.x + 1) * batch_seq_len;
    for (int idx = left_token_idx; idx < right_token_idx; idx += blockDim.x) {
      int new_idx = idx + blockIdx.x;
      new_input_ids[new_idx] = old_input_ids[idx];
    }
    if (threadIdx.x == 0) {
      new_input_ids[(blockIdx.x + 1) * (batch_seq_len + 1) - 1] = eos_id;
      old_input_ids[gridDim.x * batch_seq_len + blockIdx.x] = eos_id;
    }
    return;
  }
  int logits_token_idx_in_batch =
      blockIdx.x * logits_seq_len + logits_seq_len - 1;
  int left_logit_idx = logits_token_idx_in_batch * vocab_size + threadIdx.x;
  int right_logit_idx = (logits_token_idx_in_batch + 1) * vocab_size;

  /*
  step1. find max logit in each thread and sample from these probs with nucleus
  sampling
  */
  __shared__ float s_max_logit;
  float max_logit = CUDA_FLOAT_INF_NEG;
  for (int idx = left_logit_idx; idx < right_logit_idx; idx += blockDim.x) {
    max_logit = fmaxf(max_logit, (float)logits[idx]);
  }
  float max_logit_array[1];
  max_logit_array[0] = max_logit;
  typedef hipcub::BlockRadixSort<float, 1024, 1> BlockRadixSort;
  __shared__ typename BlockRadixSort::TempStorage sort_temp_storage;
  BlockRadixSort(sort_temp_storage).SortDescending(max_logit_array);
  float presum_max_logit_exp;
  max_logit = max_logit_array[0];

  float block_max_logit = blockReduceMax(max_logit);
  if (threadIdx.x == 0) {
    s_max_logit = block_max_logit;
  }
  __syncthreads();

  float biased_logit_exp =
      expf(fmaxf(max_logit - s_max_logit, logit_thresh_min));

  typedef hipcub::BlockScan<float, 1024> BlockScan;
  __shared__ typename BlockScan::TempStorage presum_temp_storage;
  BlockScan(presum_temp_storage)
      .InclusiveSum(biased_logit_exp, presum_max_logit_exp);

  float topp_exp_threshold;
  if (threadIdx.x == blockDim.x - 1) {
    topp_exp_threshold = p * presum_max_logit_exp;
  }
  __shared__ float s_presum_logit_exp_threshold;
  if (presum_max_logit_exp > topp_exp_threshold) {
    presum_max_logit_exp = CUDA_FLOAT_INF_NEG;
  }
  float logit_exp_threshold = blockReduceMax(presum_max_logit_exp);
  if (threadIdx.x == 0) {
    s_presum_logit_exp_threshold = logit_exp_threshold;
  }
  __syncthreads();

  __shared__ float s_logit_threshold;
  if (presum_max_logit_exp == s_presum_logit_exp_threshold) {
    s_logit_threshold = max_logit;
  }
  __syncthreads();

  /* step2 hold one logit per thread and sample
   * from them */
  float topk_exp_sum, topk_exp = CUDA_FLOAT_INF_NEG;
  int topk_tid = vocab_size;
  int test_num = 0;
  __shared__ float s_topk_exp_sum;
  for (int idx = left_logit_idx; idx < right_logit_idx; idx += blockDim.x) {
    float logit = (float)logits[idx];
    float logit_exp = expf(fmaxf(logit - s_max_logit, logit_thresh_min));
    if (logit >= s_logit_threshold) test_num++;
    if (logit >= s_logit_threshold && logit_exp > topk_exp) {
      topk_exp = logit_exp;
      topk_tid = idx - left_logit_idx + threadIdx.x;
    }
  }

  test_num = blockReduceSum(test_num);

  if (topk_tid == vocab_size) topk_exp = 0;
  topk_exp_sum = blockReduceSum(topk_exp);
  if (threadIdx.x == 0) {
    s_topk_exp_sum = topk_exp_sum;
  }
  __syncthreads();

  /* calculate cumulative probability */
  float topk_prob = topk_exp / s_topk_exp_sum;
  float prefix_sum_prob;
  BlockScan(presum_temp_storage).InclusiveSum(topk_prob, prefix_sum_prob);

  __shared__ float random_x;
  if (threadIdx.x == 0) {
    random_x = hiprand_uniform(curandstate + blockIdx.x);
  }
  __syncthreads();

  __shared__ int s_tid;
  if (threadIdx.x == 0) {
    s_tid = vocab_size;
  }
  __syncthreads();

  int threadID = threadIdx.x;
  __shared__ int s_threadID;
  __shared__ float s_max_prob;
  if (random_x > prefix_sum_prob) threadID = blockDim.x;
  threadID = blockReduceMin(threadID);
  float max_prob = blockReduceMax(topk_prob);
  if (threadIdx.x == 0) {
    s_threadID = threadID;
    s_max_prob = max_prob;
  }
  __syncthreads();
  if (threadIdx.x == s_threadID) {
    s_tid = topk_tid;
  }
  __syncthreads();

  if (s_tid == vocab_size && topk_prob == s_max_prob) {
    s_tid = topk_tid;
  }
  __syncthreads();

  /* if new sampled tid is not EOS, set unfinish TRUE */
  if (threadIdx.x == 0) {
    if (s_tid != eos_id) unfinished[0] = 1;
  }

  /* step3 copy old_input_ids to new_input_ids and add new sampled ids */
  int left_token_idx = blockIdx.x * batch_seq_len + threadIdx.x;
  int right_token_idx = (blockIdx.x + 1) * batch_seq_len;
  for (int idx = left_token_idx; idx < right_token_idx; idx += blockDim.x) {
    int new_idx = idx + blockIdx.x;
    new_input_ids[new_idx] = old_input_ids[idx];
  }
  if (threadIdx.x == 0) {
    new_input_ids[(blockIdx.x + 1) * (batch_seq_len + 1) - 1] = s_tid;
    //  save the newly sampled ids to old_input_ids for next step inputs
    old_input_ids[gridDim.x * batch_seq_len + blockIdx.x] = s_tid;
  }
}

template <typename T>
void ker_topp_sample_launcher(int batch_size, int batch_seq_len,
                              int logits_seq_len, int max_thread_per_block,
                              hipStream_t stream, const T* logits,
                              int* old_input_ids, int* new_input_ids,
                              const int* real_seq_len, const int vocab_size,
                              const float p, int* unfinished,
                              hiprandState* curandstate, int eos_id) {
  ker_topp_sample<T><<<batch_size, max_thread_per_block, 0, stream>>>(
      logits, old_input_ids, new_input_ids, real_seq_len, vocab_size,
      batch_seq_len, logits_seq_len, unfinished, p, curandstate, eos_id);
}

template void ker_topp_sample_launcher<float>(
    int batch_size, int batch_seq_len, int logits_seq_len,
    int max_thread_per_block, hipStream_t stream, const float* logits,
    int* old_input_ids, int* new_input_idx, const int* real_seq_len,
    const int vocab_size, const float p, int* unfinished,
    hiprandState* curandstate, int eos_id);

template void ker_topp_sample_launcher<__half>(
    int batch_size, int batch_seq_len, int logits_seq_len,
    int max_thread_per_block, hipStream_t stream, const __half* logits,
    int* old_input_ids, int* new_input_idx, const int* real_seq_len,
    const int vocab_size, const float p, int* unfinished,
    hiprandState* curandstate, int eos_id);

}  // namespace cuda
}  // namespace lightseq
