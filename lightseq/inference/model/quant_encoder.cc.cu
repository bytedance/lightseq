#include "hip/hip_runtime.h"
#include "quant_encoder.h"

#include "../kernels/transformerKernels.h"
#include "../kernels/embKernels.h"
#include "../kernels/transformerKernels_int8.h"
#include "cublas_helper.h"

/**
@file
Transformer encoder, composed by gemm lib and
  custom cuda kernel function
*/

namespace lightseq {
namespace cuda {

template <OperationType OpType_>
QuantEncoder<OpType_>::QuantEncoder(int max_batch_size, int *p_d_token_id,
                                    int *p_d_padding_mask,
                                    _DataType *p_d_output,
                                    const QuantTransformerWeight<OpType_> &tw,
                                    hipStream_t stream, hipblasHandle_t hd,
                                    const int *p_d_lang_id)
    : _max_batch_size(max_batch_size),
      _p_d_token_id(p_d_token_id),
      _p_d_padding_mask(p_d_padding_mask),
      _p_d_output(p_d_output),
      _p_d_lang_id(p_d_lang_id),
      _tw(tw),
      _stream(stream),
      _hd(hd),
      _p_d_src_emb_wei(tw.get_src_emb_wei()),
      _p_d_enc_wei(tw.get_enc_wei()),
      _fone((_DataType)1.f),
      _fzero((_DataType)0.f),

      _src_scaled_emb_clip_max(tw.get_src_scaled_emb_clip_max()),
      _enc_clip_max(tw.get_enc_clip_max()),
      _ione((int32_t)1),
      _izero((int32_t)0),

      _atten_scaler((_DataType)sqrt(1.f / tw._dim_per_head)),
      _max_batch_dim(max_batch_size * tw._max_step * tw._hidden_size),
      _max_thread_per_block(1024) {
  CHECK_GPU_ERROR(hipblasLtCreate(&_cublas_lt_handle));
}

/**
Compute GPU memory size needed by transformer encoder,
  to see how these memory is used, checkout init_buffer() for detail
*/
template <OperationType OpType_>
long QuantEncoder<OpType_>::compute_buffer_bytesize() {
  long sz1 = _max_batch_dim * 6 +
             _max_batch_size * _tw._head_num * _tw._max_step * _tw._max_step;
  long sz2 = _max_batch_dim + _max_batch_size * _tw._max_step * _tw._inner_size;
  return max(sz1, sz2) * sizeof(_DataType);
}

/**
Init the GPU memory pointer which point to
  the memory buffer needed by encoder.
These buffer are used during custom cuda kernel function,
  find the corresponding function to see how these buffer are used
*/
template <OperationType OpType_>
void QuantEncoder<OpType_>::init_buffer(void *pbuf) {
  _DataType *p_d_buf = reinterpret_cast<_DataType *>(pbuf);
  _p_d_qkv_projected = p_d_buf;
  _p_d_q = _p_d_qkv_projected + _max_batch_dim * 3;
  _p_d_k = _p_d_q + _max_batch_dim;
  _p_d_v = _p_d_k + _max_batch_dim;
  _p_d_c = _p_d_v + _max_batch_dim;
  _p_d_ffn_buf1 = p_d_buf;
  _p_d_ffn_buf2 = _p_d_ffn_buf1 + _max_batch_dim;
  // encoder and decoder use the same buffer to save gpu memory useage

  int max_batch_dim = _max_batch_size * _tw._max_step *
                      std::max(_tw._inner_size, _tw._hidden_size * 3);
  CHECK_GPU_ERROR(hipMalloc(&_int8_ffn_in_buf, max_batch_dim));
  CHECK_GPU_ERROR(
      hipMalloc(&_int32_ffn_out_buf, max_batch_dim * sizeof(int32_t)));
  CHECK_GPU_ERROR(
      hipMalloc(&_int8_ffn_out_buf, max_batch_dim * sizeof(int8_t)));
  _int8_p_d_enc_wei = std::vector<int8_t *>(_tw._n_enc_layer * 4);
  _scaled_ffn2_colsum = std::vector<_DataType *>(_tw._n_enc_layer);
  for (_layer_id = 0; _layer_id < _tw._n_enc_layer; _layer_id++) {
    _weight_offset = _layer_id * _tw._weight_per_enc_layer;
    CHECK_GPU_ERROR(hipMalloc(&_int8_p_d_enc_wei[_layer_id * 4],
                               _tw._hidden_size * 3 * _tw._hidden_size));
    CHECK_GPU_ERROR(hipMalloc(&_int8_p_d_enc_wei[_layer_id * 4 + 1],
                               _tw._hidden_size * _tw._hidden_size));
    CHECK_GPU_ERROR(hipMalloc(&_int8_p_d_enc_wei[_layer_id * 4 + 2],
                               _tw._hidden_size * _tw._inner_size));
    CHECK_GPU_ERROR(hipMalloc(&_int8_p_d_enc_wei[_layer_id * 4 + 3],
                               _tw._inner_size * _tw._hidden_size));

    quantize_weight(_p_d_enc_wei[_weight_offset + 2],
                    _int8_p_d_enc_wei[_layer_id * 4], _tw._hidden_size,
                    _tw._hidden_size * 3,
                    _quant_range / _enc_clip_max[_layer_id * 12], _stream,
                    _cublas_lt_handle);

    quantize_weight(_p_d_enc_wei[_weight_offset + 4],
                    _int8_p_d_enc_wei[_layer_id * 4 + 1], _tw._hidden_size,
                    _tw._hidden_size,
                    _quant_range / _enc_clip_max[_layer_id * 12 + 1], _stream,
                    _cublas_lt_handle);

    quantize_weight(_p_d_enc_wei[_weight_offset + 8],
                    _int8_p_d_enc_wei[_layer_id * 4 + 2], _tw._hidden_size,
                    _tw._inner_size,
                    _quant_range / _enc_clip_max[_layer_id * 12 + 2], _stream,
                    _cublas_lt_handle);

    quantize_weight(_p_d_enc_wei[_weight_offset + 10],
                    _int8_p_d_enc_wei[_layer_id * 4 + 3], _tw._inner_size,
                    _tw._hidden_size,
                    _quant_range / _enc_clip_max[_layer_id * 12 + 3], _stream,
                    _cublas_lt_handle);

    if (_tw._use_gelu) {
      _scaled_ffn2_colsum[_layer_id] = nullptr;
    } else {
      CHECK_GPU_ERROR(hipMalloc(&_scaled_ffn2_colsum[_layer_id],
                                 _tw._hidden_size * sizeof(_DataType)));
      float relu_scale = _enc_clip_max[_layer_id * 12 + 7] / 2;
      launch_scaled_colsum(_p_d_enc_wei[_weight_offset + 10],
                           _scaled_ffn2_colsum[_layer_id], _tw._inner_size,
                           _tw._hidden_size, relu_scale, _stream);
    }
  }

  return;
}

/**
Some requirements needed by custom cuda kernel function
*/
template <OperationType OpType_>
std::string QuantEncoder<OpType_>::check() {
  // if (_max_thread_per_block < _tw._hidden_size) {
  //   return "violate hidden_size <= max_thread_per_block";
  // }
  if (_tw._inner_size & 1) {
    return "violate inner_size % 2 = 0";
  }
  if (_tw._dim_per_head & 1) {
    return "violate dim_per_head % 2 = 0";
  }
  if (_tw._multilg_type == 0 && _p_d_src_emb_wei.size() != 4) {
    return "violate p_d_src_emb_wei.size() = 4";
  }
  if (_tw._multilg_type != 0 && _p_d_src_emb_wei.size() != 5) {
    return "violate p_d_src_emb_wei.size() = 5";
  }
  if (_p_d_enc_wei.size() != _tw._weight_per_enc_layer * _tw._n_enc_layer) {
    return "violate p_d_enc_wei.size() = weight_per_enc_layer * n_enc_layer";
  }
  if (_tw._multilg_type != 0 && _p_d_lang_id == nullptr) {
    return "lang id should not be null when multilg";
  }
  return "";
}

/**
QuantEncoder inference
*/
template <OperationType OpType_>
void QuantEncoder<OpType_>::run_one_infer(int batch_size, int batch_seq_len) {
  if (batch_size > _max_batch_size) {
    throw std::runtime_error("batch size of input greater than max_batch_size");
  }
  if (batch_seq_len > _tw._max_step) {
    throw std::runtime_error("seq len of input greater than max_step");
  }

  /* ---step1. init--- */
  _batch_size = batch_size;
  _batch_seq_len = batch_seq_len;
  _batch_token_num = batch_size * batch_seq_len;
#ifdef DEBUG_RESULT
  std::cout << "batch_size-" << batch_size << " batch_seq_len-" << batch_seq_len
            << std::endl;
  print_vec(_p_d_token_id, "batch_token_ids", batch_size * batch_seq_len);
#endif

  /* ---step2. encoder feedforward--- */
  launch_enc_emb<_DataType>(_p_d_src_emb_wei[0], _p_d_src_emb_wei[1],
                            _p_d_token_id, _p_d_output, _p_d_padding_mask,
                            _tw._padding_id, batch_size, batch_seq_len,
                            _tw._hidden_size, _stream, _p_d_src_emb_wei[4],
                            _p_d_lang_id, _tw._multilg_type);
#ifdef DEBUG_RESULT
  for (int i = 0; i < _batch_size; i++) {       // batch_id
    for (int j = 0; j < _batch_seq_len; j++) {  // token_id
      std::cout << "emb out: token-" << j << std::endl;
      print_vec(_p_d_output + i * _batch_seq_len * _tw._hidden_size +
                    j * _tw._hidden_size,
                "emb out", 10);
    }
  }  // not normal
  print_vec(_p_d_src_emb_wei[0], "token embedding weight", 10);
  print_vec(_p_d_src_emb_wei[1], "position embedding weight", 10);
#endif
  for (_layer_id = 0; _layer_id < _tw._n_enc_layer; _layer_id++) {
    _weight_offset = _layer_id * _tw._weight_per_enc_layer;
    self_attention();
    ffn_add_norm();
  }

#ifdef DEBUG_RESULT
  for (int i = 0; i < _batch_size; i++) {       // batch_id
    for (int j = 0; j < _batch_seq_len; j++) {  // token_id
      std::cout << "encoder output: token-" << j << std::endl;
      print_vec(_p_d_output + i * _batch_seq_len * _tw._hidden_size +
                    j * _tw._hidden_size,
                "encoder_output", _tw._dim_per_head);
    }
  }  // not normal
#endif
  return;
}

/**
QuantEncoder self attention
*/
template <OperationType OpType_>
void QuantEncoder<OpType_>::self_attention() {
  if (_layer_id == 0) {
    ker_norm_layer_resual_i8O_launcher<_DataType>(
        _batch_token_num, _tw._hidden_size, _stream, _p_d_output,
        _int8_ffn_in_buf, _p_d_enc_wei[_weight_offset],
        _p_d_enc_wei[_weight_offset + 1], _p_d_enc_wei[_weight_offset + 5],
        _max_thread_per_block, _quant_range / _enc_clip_max[_layer_id * 12 + 4],
        _tw._is_post_ln, true);
  }

  cublasLtMM_withAlgo_i8IO(
      _int8_ffn_out_buf, 1, _batch_token_num, _tw._hidden_size * 3,
      _tw._hidden_size, 0, 0, 0,
      _enc_clip_max[_layer_id * 12] * _enc_clip_max[_layer_id * 12 + 4] /
          (_enc_clip_max[_layer_id * 12 + 8] * _quant_range),
      _int8_ffn_in_buf, _int8_p_d_enc_wei[_layer_id * 4], _cublas_lt_handle,
      _stream, false);

  // get q, k, v by split and reshape qkv

  ker_arrange_encself_qkv_i8I_launcher<_DataType>(
      _batch_token_num, _tw._hidden_size, _stream, _int8_ffn_out_buf,
      _p_d_enc_wei[_weight_offset + 3], _p_d_q, _max_batch_dim, _batch_seq_len,
      _tw._dim_per_head, _tw._head_num, _max_thread_per_block,
      _enc_clip_max[_layer_id * 12 + 8] / _quant_range, true);

  /* ---step 2. correlation = q * k, perform softmax on correlation--- */
  CHECK_GPU_ERROR(hipblasGemmStridedBatchedEx(
      _hd, HIPBLAS_OP_T, HIPBLAS_OP_N, _batch_seq_len, _batch_seq_len,
      _tw._dim_per_head, &_atten_scaler, _p_d_k, _AType, _tw._dim_per_head,
      _batch_seq_len * _tw._dim_per_head, _p_d_q, _BType, _tw._dim_per_head,
      _batch_seq_len * _tw._dim_per_head, &_fzero, _p_d_c, _CType,
      _batch_seq_len, _batch_seq_len * _batch_seq_len,
      _batch_size * _tw._head_num, _computeType,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP));
  ker_correlation_softmax_encself_launcher<_DataType>(
      _batch_size, _batch_seq_len, _tw._head_num, _stream, _p_d_c,
      _p_d_padding_mask);

  /* ---step 3. new_q = correlation * v--- */
  CHECK_GPU_ERROR(hipblasGemmStridedBatchedEx(
      _hd, HIPBLAS_OP_N, HIPBLAS_OP_N, _tw._dim_per_head, _batch_seq_len,
      _batch_seq_len, &_fone, _p_d_v, _AType, _tw._dim_per_head,
      _batch_seq_len * _tw._dim_per_head, _p_d_c, _BType, _batch_seq_len,
      _batch_seq_len * _batch_seq_len, &_fzero, _p_d_q, _CType,
      _tw._dim_per_head, _batch_seq_len * _tw._dim_per_head,
      _batch_size * _tw._head_num, _computeType,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP));

  // use v to save reshaped q, since they are in same size and v
  // will not be use again before the next multi-head-attention
  ker_arrange_atten_output_i8O_launcher<_DataType>(
      _batch_token_num, _tw._hidden_size, _stream, _p_d_q, _int8_ffn_in_buf,
      _batch_seq_len, _tw._dim_per_head, _tw._head_num, _max_thread_per_block,
      _quant_range / _enc_clip_max[_layer_id * 12 + 5], true);

  /* ---step 4. new_q = ori_q + new_q * output_wei--- */

  cublasLtMM_withAlgo_i8IO(
      _int8_ffn_out_buf, 1, _batch_token_num, _tw._hidden_size,
      _tw._hidden_size, 0, 0, 0,
      _enc_clip_max[_layer_id * 12 + 1] * _enc_clip_max[_layer_id * 12 + 5] /
          (_enc_clip_max[_layer_id * 12 + 9] * _quant_range),
      _int8_ffn_in_buf, _int8_p_d_enc_wei[_layer_id * 4 + 1], _cublas_lt_handle,
      _stream, false);

  ker_residual_bias_ln_i8I_i8O_launcher<_DataType>(
      _int8_ffn_out_buf, _p_d_enc_wei[_weight_offset + 6],
      _p_d_enc_wei[_weight_offset + 7], _p_d_enc_wei[_weight_offset + 11],
      _int8_ffn_in_buf, _p_d_output, _batch_token_num, _tw._hidden_size,
      _enc_clip_max[_layer_id * 12 + 9] / _quant_range,
      _quant_range / _enc_clip_max[_layer_id * 12 + 6], _max_thread_per_block,
      _stream, _tw._is_post_ln, true);

  return;
}

template <OperationType OpType_>
void QuantEncoder<OpType_>::ffn_add_norm() {
  cublasLtMM_withAlgo_i8IO(
      _int8_ffn_out_buf, 1, _batch_token_num, _tw._inner_size, _tw._hidden_size,
      0, 0, 0,
      _enc_clip_max[_layer_id * 12 + 2] * _enc_clip_max[_layer_id * 12 + 6] /
          (_enc_clip_max[_layer_id * 12 + 10] * _quant_range),
      _int8_ffn_in_buf, _int8_p_d_enc_wei[_layer_id * 4 + 2], _cublas_lt_handle,
      _stream, false);

  if (_tw._use_gelu) {
    ker_bias_gelu_i8I_i8O_launcher<_DataType>(
        _batch_token_num, _stream, _int8_ffn_out_buf, _int8_ffn_in_buf,
        _p_d_enc_wei[_weight_offset + 9], _tw._inner_size,
        _enc_clip_max[_layer_id * 12 + 10] / _quant_range,
        _quant_range / _enc_clip_max[_layer_id * 12 + 7], true);
  } else {
    ker_bias_relu_i8I_i8O_launcher<_DataType>(
        _batch_token_num, _stream, _int8_ffn_out_buf, _int8_ffn_in_buf,
        _p_d_enc_wei[_weight_offset + 9], _tw._inner_size,
        _enc_clip_max[_layer_id * 12 + 10] / _quant_range,
        _quant_range / _enc_clip_max[_layer_id * 12 + 7],
        _enc_clip_max[_layer_id * 12 + 7], true, true, true);
  }

  /* ---step 2. second ffn layer--- */
  cublasLtMM_withAlgo(_int32_ffn_out_buf, 1, _batch_token_num, _tw._hidden_size,
                      _tw._inner_size, 0, 0, 0, _int8_ffn_in_buf,
                      _int8_p_d_enc_wei[_layer_id * 4 + 3], _cublas_lt_handle,
                      _stream, false);

  const _DataType *scale_ptr, *bias_ptr, *res_bias_ptr;
  float clip_max;
  if (_layer_id == _tw._n_enc_layer - 1) {
    scale_ptr = _p_d_src_emb_wei[2];
    bias_ptr = _p_d_src_emb_wei[3];

    ker_residual_bias_ln_i32I_launcher<_DataType>(
        _int32_ffn_out_buf, scale_ptr, bias_ptr, _p_d_output, _p_d_output,
        _batch_token_num, _tw._hidden_size,
        _enc_clip_max[_layer_id * 12 + 3] * _enc_clip_max[_layer_id * 12 + 7] /
            (2 * _quant_range * _quant_range),
        _max_thread_per_block, _stream, true, _scaled_ffn2_colsum[_layer_id]);
  } else {
    scale_ptr = _p_d_enc_wei[(_layer_id + 1) * _tw._weight_per_enc_layer];
    bias_ptr = _p_d_enc_wei[(_layer_id + 1) * _tw._weight_per_enc_layer + 1];
    res_bias_ptr =
        _p_d_enc_wei[(_layer_id + 1) * _tw._weight_per_enc_layer + 5];
    clip_max = _enc_clip_max[(_layer_id + 1) * 12 + 4];

    ker_residual_bias_ln_i32I_i8O_launcher<_DataType>(
        _int32_ffn_out_buf, scale_ptr, bias_ptr, res_bias_ptr, _int8_ffn_in_buf,
        _p_d_output, _batch_token_num, _tw._hidden_size,
        _enc_clip_max[_layer_id * 12 + 3] * _enc_clip_max[_layer_id * 12 + 7] /
            (2 * _quant_range * _quant_range),
        _quant_range / clip_max, _max_thread_per_block, _stream,
        _tw._is_post_ln, true, true, _scaled_ffn2_colsum[_layer_id]);
  }

  return;
}

template class QuantEncoder<OperationType::FP16>;
template class QuantEncoder<OperationType::FP32>;

}  // namespace cuda
}  // namespace lightseq
