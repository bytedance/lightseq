#include "hip/hip_runtime.h"
#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>

#include "../model/bert_encoder.h"
#include "../proto/bert_weight.h"
#include "../tools/util.h"

#ifdef FP16_MODE
const lightseq::cuda::OperationType bert_optytpe =
    lightseq::cuda::OperationType::FP16;
#else
const lightseq::cuda::OperationType bert_optytpe =
    lightseq::cuda::OperationType::FP32;
#endif

namespace py = pybind11;

namespace lightseq {
namespace cuda {
class Bert {
 private:
  typedef OperationTypeTraits<bert_optytpe> optraits;
  BertEncoder<bert_optytpe> *bert;

  optraits::DataType *d_encoder_output_;
  int *d_input_;
  int *d_padding_mask_;
  int _max_batch_size;
  hipStream_t stream_;
  hipblasHandle_t hd_;
  void *d_buf_;
  BertWeight<bert_optytpe> tw_;

 public:
  Bert(const std::string weight_path, const int max_batch_size) {
    /* ---step1. init environment--- */
    _max_batch_size = max_batch_size;
    CHECK_GPU_ERROR(hipSetDevice(0));
    CHECK_GPU_ERROR(hipStreamCreate(&stream_));
    CHECK_GPU_ERROR(hipblasCreate(&hd_));
    CHECK_GPU_ERROR(hipblasSetStream(hd_, stream_));

    /* ---step2. load model weights into GPU memory--- */

    // saved in custom proto file
    std::string model_weights_path = weight_path;
    std::string res = tw_.initializing(model_weights_path);
    if (!res.empty()) {
      throw std::runtime_error(res);
    }

    tw_.print_model_config();

    /*
      step3. instantiate encoder and decoder, init the gpu memory buffer.
        using thrust vector to avoid manage gpu memory by hand
    */

    // register device memory for inputs and outputs
    CHECK_GPU_ERROR(
        hipMalloc(&d_input_, _max_batch_size * tw_._max_step * sizeof(int)));
    CHECK_GPU_ERROR(hipMalloc(&d_padding_mask_,
                               _max_batch_size * tw_._max_step * sizeof(int)));

    CHECK_GPU_ERROR(hipMalloc(
        &d_encoder_output_, _max_batch_size * tw_._max_step * tw_._hidden_size *
                                sizeof(optraits::DataType)));

    bert =
        new BertEncoder<bert_optytpe>(max_batch_size, d_input_, d_padding_mask_,
                                      d_encoder_output_, tw_, stream_, hd_);
    res = bert->check();
    if (!res.empty()) {
      throw std::runtime_error(res);
    }

    long buf_bytesize = bert->compute_buffer_bytesize();
    std::cout << "Bert buf_bytesize: " << buf_bytesize << std::endl;

    // encoder and decoder use the same buffer to save gpu memory useage
    CHECK_GPU_ERROR(hipMalloc(&d_buf_, (size_t)buf_bytesize));
    bert->init_buffer(d_buf_);
    CHECK_GPU_ERROR(hipStreamSynchronize(stream_));
  }

  ~Bert() {
    CHECK_GPU_ERROR(hipFree(d_input_));
    CHECK_GPU_ERROR(hipFree(d_padding_mask_));
    CHECK_GPU_ERROR(hipFree(d_encoder_output_));
    CHECK_GPU_ERROR(hipFree(d_buf_));
  }

  py::array_t<float> infer(
      py::array_t<int, py::array::c_style | py::array::forcecast> input_seq,
      py::array_t<int, py::array::c_style | py::array::forcecast> attn_mask) {
    // deal with input
    auto input_seq_out = input_seq.mutable_unchecked<2>();
    const int *input_seq_data = input_seq_out.data(0, 0);
    const int *attn_mask_data = attn_mask.data(0, 0);

    int batch_size = input_seq_out.shape(0);
    int batch_seq_len = input_seq_out.shape(1);
    if (batch_size > _max_batch_size) {
      throw std::runtime_error(
          "batch size of input greater than max_batch_size");
    }
    if (batch_seq_len > tw_._max_step) {
      throw std::runtime_error("seq len of input greater than max_step");
    }
    CHECK_GPU_ERROR(hipMemcpyAsync(d_input_, input_seq_data,
                                    sizeof(int) * input_seq_out.size(),
                                    hipMemcpyHostToDevice, stream_));
    CHECK_GPU_ERROR(hipMemcpyAsync(d_padding_mask_, attn_mask_data,
                                    sizeof(int) * attn_mask.size(),
                                    hipMemcpyHostToDevice, stream_));

    // Start inference and copy result
    bert->run_one_infer(batch_size, batch_seq_len);

    auto bert_output =
        py::array_t<float>({batch_size, batch_seq_len, tw_._hidden_size});
    float *bert_output_data = bert_output.mutable_data(0, 0, 0);
    std::vector<optraits::DataType> h_bert_out(bert_output.size());

    CHECK_GPU_ERROR(
        hipMemcpyAsync(h_bert_out.data(), d_encoder_output_,
                        sizeof(optraits::DataType) * bert_output.size(),
                        hipMemcpyDeviceToHost, stream_));
    CHECK_GPU_ERROR(hipStreamSynchronize(stream_));

    for (auto i = 0; i < h_bert_out.size(); i++) {
      float data;
      if (bert_optytpe == OperationType::FP16) {
        data = __half2float(h_bert_out[i]);
      } else {
        data = h_bert_out[i];
      }
      bert_output_data[i] = data;
    }
    return bert_output;
  };
};
}  // namespace cuda
}  // namespace lightseq
